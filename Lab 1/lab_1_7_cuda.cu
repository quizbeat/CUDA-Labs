#include "hip/hip_runtime.h"
//
//  main.cpp
//  matrix-test
//
//  Created by Nikita Makarov on 22/03/16.
//  Copyright © 2016 Nikita Makarov. All rights reserved.
//

#include <iostream>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <iomanip>

using namespace std;

#define CSC(call) {														\
    hipError_t err = call;												\
    if(err != hipSuccess) {											\
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	\
            __FILE__, __LINE__, hipGetErrorString(err));				\
        exit(1);														\
    }																	\
} while (0)

const double eps = 10e-7;

__host__ void print_matrix(double **M, int n, int m) {
    cout.setf(ios::scientific);
    cout.precision(10);
    for (int i = 0; i < n; i++) {
        cout << M[i][0];
        for (int j = 1; j < m; j++) {
            cout << " " << M[i][j];
        }
        cout << endl;
    }
}

// j is a pointer to max value row on device
__global__ void swap_rows(int *prm, int i, int *j) {
    int temp = prm[i];
    prm[i] = prm[*j];
    prm[*j] = temp;
}

// inits given array with numbers from 0 to n - 1
__global__ void initSequence(double *seq, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    for ( ; index < n; index += offset) {
        seq[index] = index;
    }
}

// finds max value row beginnig from row to n, updates max_value_row pointer
__global__ void findMaxValueRow(double *M, int *prm, int row, int col, int n, int *max_value_row) {
    *max_value_row = row;
    for (int i = row + 1; i < n; i++) {
        int index_current = index_for_A(prm[i], col, n, m, k);
        int index_max = index_for_A(prm[max_value_row], col, n, m, k);
        if (fabs(M_host[index_current]) > fabs(M_host[index_max])) {
            max_value_row = i;
        }
    }
}

// updates rows from row_start to n, columns from col_start to (m + k)
// concurrently updates all columns ??
__global__ void updateRowsBelow(double *M, int *prm, int row, int col, int n, int m, int k) {
    int row_index = 0;

    int pivot_index = index_for_A(prm[row], col, n, m, k);
    double pivot = M[pivot_index];

    for (int i = row + 1; i < n; i++) {

        int factor_numerator_index = index_for_A(prm[i], col, n, m, k);
        int factor_denominator_index = index_for_A(prm[row], col, n, m, k);
        double factor = -M[factor_numerator_index] / M[factor_denominator_index];

        int column_index = col + blockIdx.x * blockDim.x + threadIdx.x; /// !!!!!!
        int offset = gridDim.x * blockDim.x;

        for ( ; column_index < (m + k); column_index += offset) {
            int target_item_index = index_for_A(prm[i], column_index, n, m, k);
            int pivot_item_index = index_for_A(prm[row], column_index, n, m, k);
            M[target_item_index] += M[pivot_item_index] * factor;
        }
    }

    x_index[row] = col;
}

__global__ void backSubstitution(double *M, double *X, double *x_index, int row, int n, int m, int k) {
    int X_column_index = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;

    for ( ; X_column_index < k; X_column_index += offset) {

        for (int i = row; i >= 0; i--) {

            int index = x_index[i];
            double sum = 0.0;

            for (int j = index + 1; j < m; j++) {
                int A_index = index_for_A(prm[i], j, n, m, k);
                int X_index = index_for_X(j, X_column_index, n, m, k);
                sum += M[A_index] * X[X_index];
            }

            int A_index = index_for_A(prm[i], index, n, m, k);
            int X_target_index = index_for_X(index, X_column_index, n, m, k);

            if (fabs(A[prm[i]][index]) > eps) {
                int B_index = index_for_B(prm[i], X_column_index, n, m, k);
                X[X_target_index] = (M[B_index] - sum) / M[A_index];
            } else {
                X[X_target_index] = 0.0;
            }
        }
    }
    __syncthreads();
}

inline int index_for_A(int i, int j, int n, int m, int k) {
    return (i * (m + k)) + j;
}

inline int index_for_B(int i, int j, int n, int m, int k) {
    return (i * (m + k)) + m + j;
}

inline int index_for_AB(int i, int j, int n, int m, int k) {
    return (i * (m + k)) + j;
}

inline int index_for_X(int i, int j, int n, int m, int k) {
    return (i * m) + j;
}

int main() {

    int n, m, k;
    cin >> n >> m >> k;

    // Solving next equation: A * X = B
    // Let's M = [A|B]
    // M = [{A_row_1}{B_row_1},...,{A_row_n},{B_row_n}]

    int M_size = (n * m) + (n * k);

    double *M_host = (double *)malloc(M_size * sizeof(double));

    // read matrix A
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            int index = index_for_A(i, j, n, m, k);
            cin >> M_host[index];
        }
    }

    // read matrix B
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < k; j++) {
            int index = index_for_B(i, j, n, m, k);
            cin >> M_host[index];
        }
    }

    // X = [{X_row_1},...,{X_row_m}]

    int X_size = m * k;

    double *X_host = (double *)malloc(X_size * sizeof(double));

    // init matrix X
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            int index = index_for_X(i, j, n, m, k);
            X_host[index] = 0.0;
        }
    }

    // Begin solving equation

    double *M_device; // matrix M on device
    double *X_device; // matrix X on device

    // alloc memory on device for matrix M
    CSC(hipMalloc((void **)&M_device, M_size * sizeof(double)));
    // CSC(hipMalloc((void **)&X_device, X_size * sizeof(double)));

    // copy matrix data
    CSC(hipMemcpy(M_device, M_host, M_size * sizeof(double), hipMemcpyHostToDevice));

    int *prm_host;   // rows permutations array on host
    int *prm_device; // rows permutations array on device

    // alloc memory for permutations array
    prm_host = (int *)malloc(n * sizeof(int));
    CSC(hipMalloc((void **)&prm_device, n * sizeof(int)));

    // init permutations array on device
    initSequence <<<32, 32>>> (prm_device, n);

    int *x_index_host;   // array with indexes for diagonal elements on host
    int *x_index_device; // array with indexes for diagonal elements on device

    // alloc memory for array with indexes
    x_index_host = (int *)malloc(n * sizeof(int));
    CSC(hipMalloc((void **)&x_index_device, n * sizeof(int)));

    // init indexes on device
    initSequence <<<32, 32>>> (x_index_device, n);

    int row = 0; // current row
    int col = 0; // current column

    int *max_value_row_device; // pointer to max value row on device
    CSC(hipMalloc((void **)&max_value_row_device, sizeof(int)));

    for ( ; row < n && col < m; row++, col++) {

        // find row with max value
        findMaxValueRow <<<1, 1>>> (M_device, prm_device, row, col, n, max_value_row_device);

        // swap rows on device
        swap_rows <<<1, 1>>> (prm_device, row, max_value_row_device);

        // copy pivot value from device to host
        double M_pivot_host;
        CSC(hipMemcpy(&M_pivot_host, M_device[prm[*max_value_row_device]], sizeof(double), hipMemcpyDeviceToHost));

        if (fabs(M_pivot_host) > eps) { // non-zero pivot value
            updateRowsBelow <<<32, 32>>> (M_device, prm_device, row, col, n, m, k);
        } else {
            row--; // need to perform next iter on current row
        }
    }

    // last row position fix
    if (row == n || col == m) {
        row--;
    }

    // perform Gauss back substitution
    backSubstitution <<<32, 32>>> (M_device, X_device, x_index_device, row, n, m, k);

    // copy matrix X from device to host
    CSC(hipMemcpy(X_host, X_device, X_size * sizeof(double), hipMemcpyDeviceToHost);

    // print X matrix
    print_matrix(X_host, m, k);

    // done.
    return 0;
}
