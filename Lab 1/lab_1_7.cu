#include "hip/hip_runtime.h"
//
//  main.cpp
//  matrix-test
//
//  Created by Nikita Makarov on 22/03/16.
//  Copyright © 2016 Nikita Makarov. All rights reserved.
//

#include <iostream>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <iomanip>

using namespace std;

#define CSC(call) {														\
    hipError_t err = call;												\
    if(err != hipSuccess) {											\
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	\
            __FILE__, __LINE__, hipGetErrorString(err));				\
        exit(1);														\
    }																	\
} while (0)

const double eps = 10e-7;

#define index_for_A(i, j, n, m, k) ((i * (m + k)) + j)
#define index_for_B(i, j, n, m, k) ((i * (m + k)) + m + j)
#define index_for_X(i, j, n, m, k) ((i * k) + j)

__host__ void print_matrix(double *M, int n, int m, int k) {
    cout.setf(ios::scientific);
    cout.precision(10);
    for (int i = 0; i < m; i++) {
        int index = index_for_X(i, 0, n, m, k);
        cout << M[index];
        for (int j = 1; j < k; j++) {
            index = index_for_X(i, j, n, m, k);
            cout << " " << M[index];
        }
        cout << endl;
    }
}

// j is a pointer to max value row on device
__global__ void swap_rows(int *prm, int i, int *j) {
    int temp = prm[i];
    prm[i] = prm[*j];
    prm[*j] = temp;
}

// inits given array with numbers from 0 to n - 1
__global__ void initSequence(int *seq, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    for ( ; index < n; index += offset) {
        seq[index] = index;
    }
}

// finds max value row beginnig from row to n, updates max_value_row pointer
__global__ void findMaxValueRow(double *M, int *prm, int row, int col, int n, int m, int k, int *max_value_row) {
    *max_value_row = row;
    for (int i = row + 1; i < n; i++) {
        int index_current = index_for_A(prm[i], col, n, m, k);
        int index_max = index_for_A(prm[*max_value_row], col, n, m, k);
        if (fabs(M[index_current]) > fabs(M[index_max])) {
            *max_value_row = i;
        }
    }
}

// updates rows from row_start to n, columns from col_start to (m + k)
// concurrently updates all columns ??
__global__ void updateRowsBelow(double *M, int *prm, int *x_index, int row, int col, int n, int m, int k) {

    for (int i = row + 1; i < n; i++) {

        int factor_numerator_index = index_for_A(prm[i], col, n, m, k);
        int factor_denominator_index = index_for_A(prm[row], col, n, m, k);  // reusing every iter ??
        double factor = -M[factor_numerator_index] / M[factor_denominator_index];

        int column_index = col + blockIdx.x * blockDim.x + threadIdx.x; /// !!!!!!
        int offset = gridDim.x * blockDim.x;

        for ( ; column_index < (m + k); column_index += offset) {
            int target_item_index = index_for_A(prm[i], column_index, n, m, k);
            int pivot_item_index = index_for_A(prm[row], column_index, n, m, k);
            M[target_item_index] += M[pivot_item_index] * factor;
        }
    }

#ifdef DEBUG
    printf("----------- setting x_index[%d] = %d\n", row, col);
#endif

    x_index[row] = col;
}

__global__ void backSubstitution(double *M, double *X, int *prm, int *x_index, int row, int n, int m, int k) {
    int X_column_index = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;

#ifdef DEBUG
    // printf(">>> backSubstitution: X_column_index = %d\n", X_column_index);
    // printf(">>> backSubstitution: offset = %d\n", offset);
    // printf(">>> backSubstitution: initial row = %d\n", row);

    printf("\n>>>>> permutations array: \n");
    for (int p = 0; p < n; p++) {
        printf("%d ", prm[p]);
    }
    printf("\n\n");

    printf("\n>>>>> x_index array: \n");
    for (int p = 0; p < n; p++) {
        printf("%d ", x_index[p]);
    }
    printf("\n\n");

#endif

    for ( ; X_column_index < k; X_column_index += offset) {

#ifdef DEBUG
    printf("\n\n>>> backSubstitution for X column [%d]\n", X_column_index);
#endif

        for (int i = row; i >= 0; i--) {

            int index = x_index[i];
            double sum = 0.0;

#ifdef DEBUG
            printf("\n\n\n     Calculating sum of known x values on row [%d]\n", i);
            printf("       Sum: ");
#endif

            for (int j = index + 1; j < m; j++) {
                int A_index = index_for_A(prm[i], j, n, m, k);
                int X_index = index_for_X(j, X_column_index, n, m, k);
                sum += M[A_index] * X[X_index];
#ifdef DEBUG
                printf("(%f * %f)", M[A_index], X[X_index]);
                if (j != m - 1) {
                    printf(" + ");
                }
#endif
            }

#ifdef DEBUG
            printf("\n      Sum = %f\n", sum);
#endif

            int A_index = index_for_A(prm[i], index, n, m, k);
            int X_target_index = index_for_X(index, X_column_index, n, m, k);

#ifdef DEBUG
            printf("     Current pivot value = %f\n", M[A_index]);
            printf("     Changing X element with indexes (%d, %d)\n", index, X_column_index);
#endif

            if (fabs(M[A_index]) > eps) {
                int B_index = index_for_B(prm[i], X_column_index, n, m, k);
                X[X_target_index] = (M[B_index] - sum) / M[A_index];
#ifdef DEBUG
                printf("    X item non zero, calculated from  [ (%f - %f) / %f ] = [%f]\n", M[B_index], sum, M[A_index], X[X_target_index]);
#endif
            } else {
#ifdef DEBUG
                printf("      X is zero\n");
#endif
                X[X_target_index] = 0.0;
            }

#ifdef DEBUG
            printf("Calculating column [%d] with row [%d]\n", X_column_index, i);
            for (int q = 0; q < m; q++) {
                int index = index_for_X(q, 0, n, m, k);
                printf("%f ", X[index]);
                for (int p = 1; p < k; p++) {
                    index = index_for_X(q, p, n, m, k);
                    printf(" %f", X[index]);
                }
                printf("\n");
            }
            printf("\n");
#endif
        }
    }
}

__global__ void printMatrix(double *M, int *prm, int n, int m, int k) {
    printf("------------------------ MATRIX M ---------------------------\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < (m + k); j++) {
            if (j == m) {
                printf("| ");
            }
            int index = index_for_A(prm[i], j, n, m, k);
            printf("%lf ", M[index]);
        }
        printf("\n");
    }
    printf("---------------------------------------------------------------\n\n");
}

int main() {

    int n, m, k;
    cin >> n >> m >> k;

#ifdef DEBUG
    cout << "Matrix A: " << n << " row(s), " << m << " column(s)\n";
    cout << "Matrix B: " << n << " row(s), " << k << " column(s)\n";
    cout << "Matrix X: " << m << " row(s), " << k << " column(s)\n";
#endif

    // Solving next equation: A * X = B
    // Let's M = [A|B]
    // M = [{A_row_1}{B_row_1},...,{A_row_n},{B_row_n}]

    int M_size = (n * m) + (n * k);

#ifdef DEBUG
    cout << "> alloc memory for M_host\n";
#endif

    double *M_host = (double *)malloc(M_size * sizeof(double));

    // read matrix A

#ifdef DEBUG
    cout << "> read M_host part A\n";
#endif

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            int index = index_for_A(i, j, n, m, k);
#ifdef DEBUG
            cout << "   (" << i << ", " << j << ") = " << index << "\n";
#endif
            cin >> M_host[index];
        }
    }

    // read matrix B

#ifdef DEBUG
    cout << "> read M_host part B\n";
#endif

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < k; j++) {
            int index = index_for_B(i, j, n, m, k);
#ifdef DEBUG
            cout << "   (" << i << ", " << j << ") = " << index << "\n";
#endif
            cin >> M_host[index];
        }
    }

#ifdef DEBUG
    // print matrices
    cout << "-------------------------- MATRIX M --------------------------\n";
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < (m + k); j++) {
            if (j == m) {
                cout << "| ";
            }
            int index = index_for_A(i, j, n, m, k);
            cout << M_host[index] << " ";
        }
        cout << "\n";
    }
    cout << "---------------------------------------------------------------\n\n";
#endif

    // X = [{X_row_1},...,{X_row_m}]

    int X_size = m * k;

#ifdef DEBUG
    cout << "> alloc memory for X_host\n";
#endif

    double *X_host = (double *)malloc(X_size * sizeof(double));

    // init matrix X
#ifdef DEBUG
    cout << "> init X_host\n";
#endif

    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            int index = index_for_X(i, j, n, m, k);
#ifdef DEBUG
            cout << "   (" << i << ", " << j << ") = " << index << "\n";
#endif
            X_host[index] = 0.0;
        }
    }

    // Begin solving equation

    double *M_device; // matrix M on device
    double *X_device; // matrix X on device

    // alloc memory on device for matrix M

#ifdef DEBUG
    cout << "> CUDA alloc memory for M_device\n";
    cout << ">    M_size = " << M_size << "\n";
#endif

    CSC(hipMalloc((void **)&M_device, M_size * sizeof(double)));
    hipDeviceSynchronize();

    // alloc memory on device for matrix X

#ifdef DEBUG
    cout << "> CUDA alloc memory for X_device\n";
#endif

    CSC(hipMalloc((void **)&X_device, X_size * sizeof(double)));
    hipDeviceSynchronize();

    // copy matrix data

#ifdef DEBUG
    cout << "> CUDA copy M_host to M_device\n";
#endif

    CSC(hipMemcpy(M_device, M_host, M_size * sizeof(double), hipMemcpyHostToDevice));
    hipDeviceSynchronize();

    CSC(hipMemcpy(X_device, X_host, X_size * sizeof(double), hipMemcpyHostToDevice));
    hipDeviceSynchronize();

    int *prm_device; // rows permutations array on device

    // alloc memory for permutations array

#ifdef DEBUG
    cout << "> CUDA alloc memory for prm_device\n";
#endif

    CSC(hipMalloc((void **)&prm_device, n * sizeof(int)));
    hipDeviceSynchronize();

    // init permutations array on device

#ifdef DEBUG
    cout << "> CUDA call init sequence for prm_device\n";
#endif

    initSequence <<<32, 32>>> (prm_device, n);
    hipDeviceSynchronize();

#ifdef DEBUG
    // check permutations init
    int *prm_host = (int *)malloc(n * sizeof(int));
    CSC(hipMemcpy(prm_host, prm_device, n * sizeof(int), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    cout << "\n--------------------- PERMUTATIONS ARRAY --------------------\n";
    for (int i = 0; i < n; i++) {
        cout << prm_host[i] << " ";
    }
    cout << "\n-------------------------------------------------------------\n\n";
#endif

    int *x_index_device; // array with indexes for diagonal elements on device

    // alloc memory for array with indexes
    CSC(hipMalloc((void **)&x_index_device, n * sizeof(int)));
    hipDeviceSynchronize();

    // init indexes on device
    initSequence <<<32, 32>>> (x_index_device, n);
    hipDeviceSynchronize();

#ifdef DEBUG
    int *x_index_host = (int *)malloc(n * sizeof(int));
    CSC(hipMemcpy(x_index_host, x_index_device, n * sizeof(int), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    cout << "---------------------- x_index after init----------------------\n";
    for (int i = 0; i < n; i++) {
        cout << x_index_host[i] << " ";
    }
    cout << "\n-------------------------------------------------------------\n";
#endif

    int row = 0; // current row
    int col = 0; // current column

    int *max_value_row_device; // pointer to max value row on device
    CSC(hipMalloc((void **)&max_value_row_device, sizeof(int)));
    hipDeviceSynchronize();

    for ( ; row < n && col < m; row++, col++) {

        // find row with max value
        findMaxValueRow <<<1, 1>>> (M_device, prm_device, row, col, n, m, k, max_value_row_device);
        hipDeviceSynchronize();

#ifdef DEBUG
        cout << "> Matrix M before swap rows\n";
        printMatrix <<<1, 1>>> (M_device, prm_device, n, m, k);
        hipDeviceSynchronize();
#endif

        // swap rows on device
        swap_rows <<<1, 1>>> (prm_device, row, max_value_row_device);
        hipDeviceSynchronize();

#ifdef DEBUG
        cout << "> Matrix M after swap rows\n";
        printMatrix <<<1, 1>>> (M_device, prm_device, n, m, k);
        hipDeviceSynchronize();
#endif

        // copy pivot value from device to host
        // int max_value_row_host;
        // CSC(hipMemcpy(&max_value_row_host, max_value_row_device, sizeof(int), hipMemcpyDeviceToHost));
        // hipDeviceSynchronize();

        int pivot_index_prm;
        CSC(hipMemcpy(&pivot_index_prm, &prm_device[row], sizeof(int), hipMemcpyDeviceToHost));
        hipDeviceSynchronize();

        double M_pivot_host;
        int M_pivot_host_index = index_for_A(pivot_index_prm, col, n, m, k);
        CSC(hipMemcpy(&M_pivot_host, &M_device[M_pivot_host_index], sizeof(double), hipMemcpyDeviceToHost));
        hipDeviceSynchronize();

#ifdef DEBUG
        printf("\n>> Position: row = %d, col = %d\n", row, col);
        printf(">> Pivot value = %f\n\n", M_pivot_host);
#endif

        if (fabs(M_pivot_host) > eps) { // non-zero pivot value
            updateRowsBelow <<<32, 32>>> (M_device, prm_device, x_index_device, row, col, n, m, k);
            hipDeviceSynchronize();
        } else {
            row--; // need to perform next iter on current row
        }

#ifdef DEBUG
        cout << "> Matrix M after updating rows\n";
        printMatrix <<<1, 1>>> (M_device, prm_device, n, m, k);
        hipDeviceSynchronize();
#endif
    }

    // last row position fix
    if (row == n || col == m) {
        row--;
    }

    // perform Gauss back substitution
    backSubstitution <<<32, 32>>> (M_device, X_device, prm_device, x_index_device, row, n, m, k);
    hipDeviceSynchronize();

    // copy matrix X from device to host
    CSC(hipMemcpy(X_host, X_device, X_size * sizeof(double), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();

    // print X matrix
    print_matrix(X_host, n, m, k);

    // free device memory
    CSC(hipFree(M_device));
    CSC(hipFree(X_device));
    CSC(hipFree(prm_device));
    CSC(hipFree(x_index_device));

    // free host memory
    free(M_host);
    free(X_host);

    // done.
    return 0;
}
