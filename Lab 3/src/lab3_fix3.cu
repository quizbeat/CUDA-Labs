#include "hip/hip_runtime.h"
//
//  lab3.cu
//  CUDA-Lab-3
//
//  Created by Nikita Makarov on 07/05/16.
//  Copyright © 2016 Nikita Makarov. All rights reserved.
//

#include <iostream>
#include <stdio.h>
#include <limits>
#include <cfloat>
#include <math.h>
#include <time.h>
#include <fstream>

#define CSC(call) {														\
    hipError_t err = call;												\
    if(err != hipSuccess) {											\
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	\
            __FILE__, __LINE__, hipGetErrorString(err));				\
        exit(1);														\
    }																	\
} while (0)


#define MIN(a, b) (a < b ? a : b)
#define MAX(a, b) (a > b ? a : b)

#define EPS 1e-7

// gpu defined properties
#define GRID_SIZE  32
#define BLOCK_SIZE 32

#define GRID_SIZE_REDUCE  1024
#define BLOCK_SIZE_REDUCE 1024

#define GRID_SIZE_SCAN  512
#define BLOCK_SIZE_SCAN 512

#define GRID_SIZE_HISTOGRAM  512
#define BLOCK_SIZE_HISTOGRAM 512

// some hacks
#define LOG_NUM_BANKS 5 // for 32
#define CONFLICT_FREE_OFFSET(i) ((i) >> LOG_NUM_BANKS)

// sort properties
#define BUCKET_SIZE 1024
#define SPLIT_SIZE  32

#define INDEX_FROM_FLOAT_VALUE(value,min,max,count) (int)((value-min)/(max-min)*(count-1))
#define SWAP_FLOATS(a,b) {float t = a; a = b; b = t;}



// recursion depth control
int depth = 0;
int max_depth = 0;

void depth_inc() {
    depth++;
    if (depth > max_depth) {
        max_depth = depth;
    }
}

void depth_dec() {
    depth--;
}

void print_depth_bar() {
    for (int i = 0; i < depth; i++) {
        printf("__");
    }
}

void print_depth_space() {
    for (int i = 0; i < depth; i++) {
        printf("  ");
    }
}




// =============================================================================
//                                   PRINT
// =============================================================================

void print_array(float *data, int size) {
    for (int i = 0; i < size; i++) {
        printf("%f ", data[i]);
    }
    printf("\n");
}

void print_array(int *data, int size) {
    for (int i = 0; i < size; i++) {
        printf("%d ", data[i]);
    }
    printf("\n");
}




// =============================================================================
//                              DATA READ/WRITE
// =============================================================================

float *read_data(int *n) {
    fread(n, sizeof(int), 1, stdin);
    float *data = (float *)malloc(*n * sizeof(float));
    fread(data, sizeof(float), *n, stdin);
    return data;
}

float *read_data_as_plain_text(int *n) {
    scanf("%d", n);
    float *data = (float *)malloc(*n * sizeof(float));
    int count = 0;
    for (int i = 0; i < *n; i++) {
        scanf("%f", &data[i]);
        count++;
    }
#ifdef DEBUG
    printf("data count: %d\n", count);
#endif

    return data;
}

void write_data(float *data, int n) {
    fwrite(data, sizeof(float), n, stdout);
}

void write_data_with_size(float *data, int n) {
    fwrite(&n, sizeof(int), 1, stdout);
    fwrite(data, sizeof(float), n, stdout);
}




// =============================================================================
//                                   REDUCE
// =============================================================================

__global__ void gpuReduceMaxFloat(float *data, int n, float *result) {

    __shared__ float shared_data[2 * BLOCK_SIZE_REDUCE];

    int global_thread_id = 2 * blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id = threadIdx.x;

    if (global_thread_id + blockDim.x < n) {
        shared_data[thread_id] = MAX(data[global_thread_id], data[global_thread_id + blockDim.x]);
    } else if (global_thread_id < n) {
        shared_data[thread_id] = data[global_thread_id];
    } else {
        shared_data[thread_id] = data[0]; // just dummy
    }

    __syncthreads();;

    // reduction process
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (thread_id < i) {
            shared_data[thread_id] = MAX(shared_data[thread_id], shared_data[thread_id + i]);
        }
        __syncthreads();
    }

    // write result to global memory
    if (thread_id == 0) {
        result[blockIdx.x] = shared_data[0];
    }
}

__global__ void gpuReduceMinFloat(float *data, int n, float *result) {

    __shared__ float shared_data[2 * BLOCK_SIZE_REDUCE];

    int global_thread_id = 2 * blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id = threadIdx.x;

    if (global_thread_id + blockDim.x < n) {
        shared_data[thread_id] = MIN(data[global_thread_id], data[global_thread_id + blockDim.x]);
    } else if (global_thread_id < n) {
        shared_data[thread_id] = data[global_thread_id];
    } else {
        shared_data[thread_id] = data[0]; // just dummy
    }

    __syncthreads();;

    // reduction process
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (thread_id < i) {
            shared_data[thread_id] = MIN(shared_data[thread_id], shared_data[thread_id + i]);
        }
        __syncthreads();
    }

    // write result to global memory
    if (thread_id == 0) {
        result[blockIdx.x] = shared_data[0];
    }
}

__host__ void recursive_gpu_reduce_max(float *data_device, int n, float *result_host) {
    int gridSize = (n / (2 * BLOCK_SIZE_REDUCE)) + 1;
    int blockSize = BLOCK_SIZE_REDUCE;

    // printf("gridSize = %d\n", gridSize);

    float *result_device = NULL;
    CSC(hipMalloc((void **)&result_device, gridSize * sizeof(float)));
    CSC(hipGetLastError());

    if (result_device == NULL) {
        printf("YOLO! NULL MEMORY DETECTED!\n");
    }

    gpuReduceMaxFloat <<<gridSize, blockSize>>> (data_device, n, result_device);
    CSC(hipGetLastError());

    if (gridSize > 1) {
        recursive_gpu_reduce_max(result_device, gridSize, result_host);
    } else {
        CSC(hipMemcpy(result_host, result_device, sizeof(float), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());
    }
    CSC(hipFree(result_device));
    CSC(hipGetLastError());
}

__host__ void recursive_gpu_reduce_min(float *data_device, int n, float *result_host) {
    int gridSize = (n / (2 * BLOCK_SIZE_REDUCE)) + 1;
    int blockSize = BLOCK_SIZE_REDUCE;

    // printf("gridSize = %d\n", gridSize);

    float *result_device = NULL;
    CSC(hipMalloc((void **)&result_device, gridSize * sizeof(float)));
    CSC(hipGetLastError());

    if (result_device == NULL) {
        print_depth_space();
        printf("YOLO! NULL MEMORY DETECTED!\n");
    }

#ifdef DEBUG
    print_depth_space();
    printf("before reduce min: n = %d, gridSize = %d, blockSize = %d\n", n, gridSize, blockSize);
#endif

    gpuReduceMinFloat <<<gridSize, blockSize>>> (data_device, n, result_device);
    CSC(hipGetLastError());

    if (gridSize > 1) {
        recursive_gpu_reduce_min(result_device, gridSize, result_host);
    } else {
        CSC(hipMemcpy(result_host, result_device, sizeof(float), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());
    }
    CSC(hipFree(result_device));
    CSC(hipGetLastError());
}




// =============================================================================
//                                    SCAN
// =============================================================================

__global__ void gpuScan(int *data, int n, int *sums, int *result) {

	__shared__ int shared_data[2 * BLOCK_SIZE_SCAN + CONFLICT_FREE_OFFSET(2 * BLOCK_SIZE_SCAN)];

	int thread_id = threadIdx.x;
	int offset = 1;

	int ai = thread_id;
	int bi = thread_id + BLOCK_SIZE_SCAN;

	int offset_A = CONFLICT_FREE_OFFSET(ai);
	int offset_B = CONFLICT_FREE_OFFSET(bi);

#ifdef DEBUG
    printf("_______ BLOCK_[%d], THREAD_[%d] : (AI_0) ai = %d\n", blockIdx.x, threadIdx.x, ai);
    printf(">>> BLOCK_[%d], THREAD_[%d] : (1) assign 0 to shared_data[%d]\n", blockIdx.x, threadIdx.x, ai);
    printf(">>> BLOCK_[%d], THREAD_[%d] : (1) assign 0 to shared_data[%d]\n", blockIdx.x, threadIdx.x, bi);
#endif

	shared_data[ai] = 0;
	shared_data[bi] = 0;

	__syncthreads();

    int data_index;

    data_index = ai + 2 * BLOCK_SIZE_SCAN * blockIdx.x;

	if (data_index < n) {
#ifdef DEBUG
        printf(">>> BLOCK_[%d], THREAD_[%d] : (2) shared_data[%d] = data[%d]\n", blockIdx.x, threadIdx.x, ai + offset_A, ai + 2 * BLOCK_SIZE_SCAN * blockIdx.x);
#endif
        shared_data[ai + offset_A] = data[data_index];
	} else {
#ifdef DEBUG
        printf(">>> BLOCK_[%d], THREAD_[%d] : (2) shared_data[%d] = 0\n", blockIdx.x, threadIdx.x, ai + offset_A);
#endif
        shared_data[ai + offset_A] = 0;
	}

    data_index = bi + 2 * BLOCK_SIZE_SCAN * blockIdx.x;

	if (data_index < n) {
#ifdef DEBUG
        printf(">>> BLOCK_[%d], THREAD_[%d] : (3) shared_data[%d] = data[%d]\n", blockIdx.x, threadIdx.x, bi + offset_B, bi + 2 * BLOCK_SIZE_SCAN * blockIdx.x);
#endif
        shared_data[bi + offset_B] = data[data_index];
	} else {
#ifdef DEBUG
        printf(">>> BLOCK_[%d], THREAD_[%d] : (3) shared_data[%d] = 0\n", blockIdx.x, threadIdx.x, bi + offset_B);
#endif
        shared_data[bi + offset_B] = 0;
	}

	for (int d = BLOCK_SIZE_SCAN; d > 0; d /= 2) {
		__syncthreads();

		if (thread_id < d) {

			int ai = offset * (2 * thread_id + 1) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);

#ifdef DEBUG
            printf("_______ BLOCK_[%d], THREAD_[%d] : (AI_1) ai = %d\n", blockIdx.x, threadIdx.x, ai);
#endif
			int bi = offset * (2 * thread_id + 2) - 1;
            bi += CONFLICT_FREE_OFFSET(bi);

#ifdef DEBUG
            printf("_______ BLOCK_[%d], THREAD_[%d] : (AI_2) ai = %d\n", blockIdx.x, threadIdx.x, ai);
            printf(">>> BLOCK_[%d], THREAD_[%d] : (4) shared_data[%d] = shared_data[%d]\n", blockIdx.x, threadIdx.x, ai, bi);
#endif
			shared_data[bi] += shared_data[ai];
		}
		offset *= 2;
	}

	if (thread_id == 0) {
		int index = 2 * BLOCK_SIZE_SCAN - 1 + CONFLICT_FREE_OFFSET(2 * BLOCK_SIZE_SCAN - 1);

#ifdef DEBUG
        printf(">>> BLOCK_[%d], THREAD_[%d] : (5) sums[%d] = shared_data[%d]\n", blockIdx.x, threadIdx.x, blockIdx.x, index);
#endif

		sums[blockIdx.x] = shared_data[index];

#ifdef DEBUG
        printf(">>> BLOCK_[%d], THREAD_[%d] : (5) shared_data[%d] = 0\n", blockIdx.x, threadIdx.x, index);
#endif

		shared_data[index] = 0;
	}

	__syncthreads();

	for (int d = 1; d < 2 * BLOCK_SIZE_SCAN; d *= 2) {

		offset /= 2;

		__syncthreads();

		if (thread_id < d) {

#ifdef DEBUG
            int __ai = offset * (2 * thread_id + 1) - 1;
            int __conf = CONFLICT_FREE_OFFSET(__ai);
            int __new_ai = __ai + __conf;
            printf("************* BLOCK_[%d], THREAD_[%d] : (XXX) d = %d, offset = %d, ai = %d, CONFLICT_FREE_OFFSET(ai) = %d, new ai = %d\n", blockIdx.x, threadIdx.x, d, offset, __ai, __conf, __new_ai);
#endif

			int ai = offset * (2 * thread_id + 1) - 1;

#ifdef DEBUG
            printf("_______ BLOCK_[%d], THREAD_[%d] : (AI_4) ai = %d\n", blockIdx.x, threadIdx.x, ai);
#endif

            ai += CONFLICT_FREE_OFFSET(ai);

#ifdef DEBUG
            printf("_______ BLOCK_[%d], THREAD_[%d] : (AI_3) ai = %d\n", blockIdx.x, threadIdx.x, ai);
#endif

			int bi = offset * (2 * thread_id + 2) - 1;
			bi += CONFLICT_FREE_OFFSET(bi);

#ifdef DEBUG
            printf(">>> BLOCK_[%d], THREAD_[%d] : (6) t = shared_data[%d]\n", blockIdx.x, threadIdx.x, ai);
#endif
            int temp = shared_data[ai];

#ifdef DEBUG
            printf(">>> BLOCK_[%d], THREAD_[%d] : (6) shared_data[%d] = shared_data[%d]\n", blockIdx.x, threadIdx.x, ai, bi);
#endif
            shared_data[ai] = shared_data[bi];

#ifdef DEBUG
            printf(">>> BLOCK_[%d], THREAD_[%d] : (6) shared_data[%d] += temp\n", blockIdx.x, threadIdx.x, bi);
#endif
            shared_data[bi] += temp;
		}
	}

	__syncthreads();

    data_index = ai + 2 * BLOCK_SIZE_SCAN * blockIdx.x;

	if (data_index < n) {
#ifdef DEBUG
        printf(">>> BLOCK_[%d], THREAD_[%d] : (7) result[%d] = shared_data[%d]\n", blockIdx.x, threadIdx.x, ai + 2 * BLOCK_SIZE_SCAN * blockIdx.x, ai + offset_A);
#endif
		result[data_index] = shared_data[ai + offset_A];
	}

    data_index = bi + 2 * BLOCK_SIZE_SCAN * blockIdx.x;

	if (data_index < n) {
#ifdef DEBUG
        printf(">>> BLOCK_[%d], THREAD_[%d] : (7) result[%d] = shared_data[%d]\n", blockIdx.x, threadIdx.x, bi + 2 * BLOCK_SIZE_SCAN * blockIdx.x, bi + offset_B);
#endif
		result[data_index] = shared_data[bi + offset_B];
	}
}


__global__ void scanDistribute(int *data, int n, int *sums) {
    int idx = blockIdx.x * 2 * BLOCK_SIZE_SCAN + threadIdx.x;
	if (idx < n){
		data[idx] += sums[blockIdx.x];
	}
}


void recursive_gpu_scan(int *data, int n, int *result) {

	int threadsPerBlock = BLOCK_SIZE_SCAN;
	int threads = BLOCK_SIZE_SCAN * 2;
	int numBlocks = n/ (2 * BLOCK_SIZE_SCAN) + 1;

	int *sums  = NULL;
	int *sums2 = NULL;

	CSC(hipMalloc((void **)&sums, numBlocks * sizeof(int)));
    CSC(hipMemset(sums, 0, numBlocks * sizeof(int)));
    CSC(hipGetLastError());

	CSC(hipMalloc((void **)&sums2, numBlocks * sizeof(int)));
    CSC(hipMemset(sums2, 0, numBlocks * sizeof(int)));
    CSC(hipGetLastError());

	gpuScan <<<numBlocks, threadsPerBlock>>> (data, n, sums, result);
    CSC(hipDeviceSynchronize());
	CSC(hipGetLastError());

	if (n >= threads) {
        recursive_gpu_scan(sums, numBlocks, sums2);
	} else {
		CSC(hipMemcpy(sums2, sums, numBlocks * sizeof(int), hipMemcpyDeviceToDevice));
        CSC(hipGetLastError());
    }

	if (numBlocks > 1) {
		dim3 blocks(numBlocks - 1, 1, 1);
		dim3 threads(2 * BLOCK_SIZE_SCAN, 1, 1);
		scanDistribute <<<blocks, threads>>> (result + 2 * BLOCK_SIZE_SCAN, n - 2 * BLOCK_SIZE_SCAN, sums2 + 1);
		CSC(hipGetLastError());
	}

	CSC(hipFree(sums));
    CSC(hipGetLastError());

	CSC(hipFree(sums2));
    CSC(hipGetLastError());
}




// =============================================================================
//                                  HISTOGRAM
// =============================================================================

// gpu histogram
__global__ void gpuHistogramCalculateSplitsSizes(float *data, int n, int *result, float min, float max, int splits_count) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    for ( ; index < n; index += offset) {
        int insertion_index = INDEX_FROM_FLOAT_VALUE(data[index], min, max, splits_count);
        atomicAdd(&(result[insertion_index]), 1);
    }
}

__global__ void gpuHistogramFillSplits(float *data_device, int n, float *splits_device,
                                       int *begin_position_for_split_device,
                                       unsigned int *current_size_of_split_device,
                                       float min, float max, int splits_count)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    for ( ; index < n; index += offset) {
        float value = data_device[index];
        int split_index = INDEX_FROM_FLOAT_VALUE(value, min, max, splits_count);
        // int current_size_of_split = atomicInc(&(current_size_of_split_device[split_index]), 1); // WARNING! POTENTIAL ERROR!
        int current_size_of_split = atomicAdd(&(current_size_of_split_device[split_index]), 1); /// ?????
        int insert_position = begin_position_for_split_device[split_index] + current_size_of_split;
        splits_device[insert_position] = value;
#ifdef DEBUG
        printf("Block_id = %d, thread_id = %d -- insert value [%f] from index [%d] to splits index [%d]\n", blockIdx.x, threadIdx.x, value, index, insert_position);
#endif
    }
}




// =============================================================================
//                                   SORT
// =============================================================================

// 1 block sorts 1 bucket
__global__ void gpuOddEvenSort(float *buckets, int n, int *begin_position_for_bucket, int *size_of_bucket) {

    int bucket_index = blockIdx.x;
    int bucket_size = size_of_bucket[bucket_index];

    if (bucket_size == -1) { // bucket already sorted
        return;
    }

#ifdef DEBUG
    printf(">>> BLOCK_[%d], THREAD_[%d] : bucket_index = %d\n", blockIdx.x, threadIdx.x, bucket_index);
#endif

    // prepare shared array for bucket
    __shared__ float shared_bucket[BUCKET_SIZE];

    int thread_id = threadIdx.x;

#ifdef DEBUG
    printf(">>> BLOCK_[%d], THREAD_[%d] : shared index [%d]\n", blockIdx.x, threadIdx.x, 2 * thread_id);
#endif
    shared_bucket[2 * thread_id    ] = FLT_MAX; // dummy for item index out of bounds

#ifdef DEBUG
    printf(">>> BLOCK_[%d], THREAD_[%d] : shared index [%d]\n", blockIdx.x, threadIdx.x, 2 * thread_id + 1);
#endif
    shared_bucket[2 * thread_id + 1] = FLT_MAX; // dummy for item index out of bounds


    __syncthreads();

    // load bucket items
    int item_index = 0;

    item_index = 2 * thread_id + begin_position_for_bucket[bucket_index];
    if (item_index - begin_position_for_bucket[bucket_index] < bucket_size) {
#ifdef DEBUG
        printf(">>> BLOCK_[%d], THREAD_[%d] : assign to shared index [%d] from buckets index [%d]\n", blockIdx.x, threadIdx.x, 2 * thread_id, item_index);
#endif
        shared_bucket[2 * thread_id] = buckets[item_index];
    }

    item_index = 2 * thread_id + 1 + begin_position_for_bucket[bucket_index];
    if (item_index - begin_position_for_bucket[bucket_index] < bucket_size) {
#ifdef DEBUG
        printf(">>> BLOCK_[%d], THREAD_[%d] : assign to shared index [%d] from buckets index [%d]\n", blockIdx.x, threadIdx.x, 2 * thread_id + 1, item_index);
#endif
        shared_bucket[2 * thread_id + 1] = buckets[item_index];
    }

    __syncthreads();

    int  odd_index_limit = BUCKET_SIZE - 1;
    int even_index_limit = BUCKET_SIZE;

    for (int i = 0; i < blockDim.x; i++) {
        item_index = 2 * thread_id + 1;
        if (item_index < odd_index_limit) {
#ifdef DEBUG
            printf(">>> BLOCK_[%d], THREAD_[%d] : comparing shared indexes [%d] and [%d]\n", blockIdx.x, threadIdx.x, item_index, item_index + 1);
#endif
            if (shared_bucket[item_index] > shared_bucket[item_index + 1]) {
                SWAP_FLOATS(shared_bucket[item_index], shared_bucket[item_index + 1]);
            }
        }
        __syncthreads();
        item_index = 2 * thread_id;
        if (thread_id < even_index_limit) {
            if (shared_bucket[item_index] > shared_bucket[item_index + 1]) {
#ifdef DEBUG
                printf(">>> BLOCK_[%d], THREAD_[%d] : comparing shared indexes [%d] and [%d]\n", blockIdx.x, threadIdx.x, item_index, item_index + 1);
#endif
                SWAP_FLOATS(shared_bucket[item_index], shared_bucket[item_index + 1]);
            }
        }
        __syncthreads();
    }

    // write result back

    item_index = 2 * thread_id + begin_position_for_bucket[bucket_index];
    if (item_index - begin_position_for_bucket[bucket_index] < bucket_size) {
#ifdef DEBUG
        printf(">>> BLOCK_[%d], THREAD_[%d] : assign to buckets index [%d] from shared index [%d]\n", blockIdx.x, threadIdx.x, 2 * thread_id + 1, item_index, 2 * thread_id);
#endif
        buckets[item_index] = shared_bucket[2 * thread_id];

    }

    item_index = 2 * thread_id + 1 + begin_position_for_bucket[bucket_index];
    if (item_index - begin_position_for_bucket[bucket_index] < bucket_size) {
#ifdef DEBUG
        printf(">>> BLOCK_[%d], THREAD_[%d] : assign to buckets index [%d] from shared index [%d]\n", blockIdx.x, threadIdx.x, 2 * thread_id + 1, item_index, 2 * thread_id + 1);
#endif
        buckets[item_index] = shared_bucket[2 * thread_id + 1];
    }
}




//
// gpu_bucket_sort description:
//  data_device -- initial array allocated for GPU usage;
//            n -- amount of items in data array;
//
__host__ void gpu_bucket_sort(float *data_device, int n) {

#ifdef DEBUG
    depth_inc();
    print_depth_space();
    printf("BEGIN SORT\n");
#endif

    // find min data value
    float min = FLT_MAX;
    recursive_gpu_reduce_min(data_device, n, &min);
    CSC(hipGetLastError());

    // find max data value
    float max = -FLT_MAX;
    recursive_gpu_reduce_max(data_device, n, &max);
    CSC(hipGetLastError());


#ifdef DEBUG
    print_depth_space();
    printf("MIN = %f, MAX = %f\n", min, max);
#endif

    // check for already sorted array
    if (fabs(min - max) < EPS) {
        depth_dec();
#ifdef DEBUG
        depth_dec();
#endif
        return;
    }

    // make splits
    int splits_count = n / SPLIT_SIZE + 1;

#ifdef DEBUG
    print_depth_space();
    printf("splits_count = %d\n", splits_count);
#endif

    // create size_of_split on gpu
    int *size_of_split_device = NULL;
    CSC(hipMalloc((void **)&size_of_split_device, splits_count * sizeof(int)));
    CSC(hipMemset(size_of_split_device, 0, splits_count * sizeof(int)));
    CSC(hipGetLastError());

    // calculate splits sizes with histogram
    gpuHistogramCalculateSplitsSizes <<<GRID_SIZE_HISTOGRAM, BLOCK_SIZE_HISTOGRAM>>> (data_device, n, size_of_split_device, min, max, splits_count);
    CSC(hipGetLastError());


#ifdef DEBUG // check size_of_split array

    int *size_of_split = (int *)malloc(splits_count * sizeof(int));
    memset(size_of_split, 0, splits_count * sizeof(int));

    CSC(hipMemcpy(size_of_split, size_of_split_device, splits_count * sizeof(int), hipMemcpyDeviceToHost));
    CSC(hipGetLastError());

    print_depth_space();
    printf("size_of_split: ");
    print_array(size_of_split, splits_count);

    free(size_of_split);

#endif

    // calculate splits begin position with scan
    int *begin_position_for_split_device = NULL;
    CSC(hipMalloc((void **)&begin_position_for_split_device, splits_count * sizeof(int)));
    CSC(hipGetLastError());

    recursive_gpu_scan(size_of_split_device, splits_count, begin_position_for_split_device);
    CSC(hipGetLastError());


#ifdef DEBUG // check begin_position_for_split

    int *begin_position_for_split = (int *)malloc(splits_count * sizeof(int));
    CSC(hipMemcpy(begin_position_for_split, begin_position_for_split_device, splits_count * sizeof(int), hipMemcpyDeviceToHost));
    CSC(hipGetLastError());

    print_depth_space();
    printf("begin_position_for_split: ");
    print_array(begin_position_for_split, splits_count);

    free(begin_position_for_split);

#endif

    unsigned int *current_size_of_split_device = NULL;
    CSC(hipMalloc((void **)&current_size_of_split_device, splits_count * sizeof(unsigned int)));
    CSC(hipGetLastError());

    CSC(hipMemset(current_size_of_split_device, 0, splits_count * sizeof(unsigned int)));
    CSC(hipGetLastError());

    // create splits array
    float *splits_device = NULL;
    CSC(hipMalloc((void **)&splits_device, n * sizeof(float)));
    CSC(hipGetLastError());

    // fill splits with histogram
    gpuHistogramFillSplits <<<GRID_SIZE_HISTOGRAM, BLOCK_SIZE_HISTOGRAM>>> (data_device, n, splits_device,
                                                                            begin_position_for_split_device,
                                                                            current_size_of_split_device,
                                                                            min, max, splits_count);
    CSC(hipGetLastError());


#ifdef DEBUG // check splits array

    float *splits = (float *)malloc(n * sizeof(float));
    CSC(hipMemcpy(splits, splits_device, n * sizeof(float), hipMemcpyDeviceToHost));
    CSC(hipGetLastError());

    print_depth_space();
    printf("splits: ");
    print_array(splits, n);

    free(splits);

#endif


    // make buckets
    int buckets_count = splits_count;
    int *size_of_bucket = (int *)malloc(buckets_count * sizeof(int));
    memset(size_of_bucket, 0, buckets_count * sizeof(int));

    int *begin_position_for_bucket = (int *)malloc(buckets_count * sizeof(int));

    int bucket_index = 0;

    for (int split_index = 0; split_index < splits_count; split_index++) {

        int split_size = 0;
        CSC(hipMemcpy(&split_size, &(size_of_split_device[split_index]), sizeof(int), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());

        int split_begin_position = 0;
        CSC(hipMemcpy(&split_begin_position, &(begin_position_for_split_device[split_index]), sizeof(int), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());

#ifdef DEBUG
        print_depth_space();
        printf("split_size = %d\n", split_size);
        print_depth_space();
        printf("split_begin_position = %d\n", split_begin_position);
#endif

        if (split_size > BUCKET_SIZE) {

            bucket_index++;

            // sort current split
            float *split = &(splits_device[split_begin_position]); // gpu pointer
            gpu_bucket_sort(split, split_size);

            // remember split as bucket
            begin_position_for_bucket[bucket_index] = split_begin_position; // ????
            // size_of_bucket[bucket_index] = split_size; // ????
            size_of_bucket[bucket_index] = -1; // -1 indicates that bucket already sorted
            bucket_index++;

        } else {

            int current_bucket_remaining_capacity = BUCKET_SIZE - size_of_bucket[bucket_index];
            if (split_size <= current_bucket_remaining_capacity) {
                // insert split to current bucket
                if (current_bucket_remaining_capacity == BUCKET_SIZE) {
                    begin_position_for_bucket[bucket_index] = split_begin_position;
                }
                size_of_bucket[bucket_index] += split_size;

            } else {
                // insert split to next bucket
                bucket_index++;
                begin_position_for_bucket[bucket_index] = split_begin_position;
                size_of_bucket[bucket_index] = split_size;
            }
        }
    }

    CSC(hipFree(size_of_split_device));
    CSC(hipFree(begin_position_for_split_device));
    CSC(hipFree(current_size_of_split_device));

    // determine correct buckets count
    if (size_of_bucket[bucket_index] == 0) {
        buckets_count = bucket_index;
    } else {
        buckets_count = bucket_index + 1;
    }

    // sort buckets

    int *begin_position_for_bucket_device = NULL;
    CSC(hipMalloc((void **)&begin_position_for_bucket_device, buckets_count * sizeof(int)));
    CSC(hipGetLastError());
    CSC(hipMemcpy(begin_position_for_bucket_device, begin_position_for_bucket, buckets_count * sizeof(int), hipMemcpyHostToDevice));
    CSC(hipGetLastError());

    int *size_of_bucket_device = NULL;
    CSC(hipMalloc((void **)&size_of_bucket_device, buckets_count * sizeof(int)));
    CSC(hipGetLastError());
    CSC(hipMemcpy(size_of_bucket_device, size_of_bucket, buckets_count * sizeof(int), hipMemcpyHostToDevice));
    CSC(hipGetLastError());

    dim3 sortBlocks(buckets_count, 1, 1);
    dim3 sortThreads(BUCKET_SIZE / 2, 1, 1);

    gpuOddEvenSort <<<sortBlocks, sortThreads>>> (splits_device, n, begin_position_for_bucket_device, size_of_bucket_device);
    CSC(hipDeviceSynchronize());
    CSC(hipGetLastError());

    CSC(hipMemcpy(data_device, splits_device, n * sizeof(float), hipMemcpyDeviceToDevice));
    CSC(hipGetLastError());

    CSC(hipFree(splits_device));
    CSC(hipFree(begin_position_for_bucket_device));
    CSC(hipFree(size_of_bucket_device));
    CSC(hipGetLastError());

    free(size_of_bucket);
    free(begin_position_for_bucket);

#ifdef DEBUG
    print_depth_space();
    printf("END SORT\n");
    depth_dec();
#endif

}


//
// bucket_sort description:
//  data -- initial array allocated for CPU usage;
//     n -- amount of items in data array;
//
__host__ void bucket_sort(float *data, int n) {
    // prepare device data
    float *data_device = NULL;
    CSC(hipMalloc((void **)&data_device, n * sizeof(float)));
    CSC(hipMemcpy(data_device, data, n * sizeof(float), hipMemcpyHostToDevice));
    CSC(hipGetLastError());

    gpu_bucket_sort(data_device, n);

    CSC(hipMemcpy(data, data_device, n * sizeof(float), hipMemcpyDeviceToHost));
    CSC(hipGetLastError());
}


// =============================================================================
//                                  MAIN
// =============================================================================

bool sorted(float *data, int n) {
    for (int i = 1; i < n; i++) {
        if (data[i] < data[i - 1]) {
            return false;
        }
    }
    return true;
}


int main() {

    int n = 0;
    float *data = read_data_as_plain_text(&n);
    // float *data = read_data(&n);

    if (n == 0) {
        free(data);
        return 0;
    }

    bucket_sort(data, n);

    // print_array(data, n);

    if (sorted(data, n)) {
        printf("--\nStatus: OK\n");
    } else {
        printf("--\nStatus: WA\n");
    }

    free(data);

    return 0;
}
