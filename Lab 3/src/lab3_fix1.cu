//
//  lab3.cu
//  CUDA-Lab-3
//
//  Created by Nikita Makarov on 07/05/16.
//  Copyright © 2016 Nikita Makarov. All rights reserved.
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <limits>
#include <cfloat>
#include <math.h>
#include <time.h>
#include <fstream>

#define CSC(call) {														\
    hipError_t err = call;												\
    if(err != hipSuccess) {											\
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	\
            __FILE__, __LINE__, hipGetErrorString(err));				\
        exit(1);														\
    }																	\
} while (0)


#define MIN(a, b) (a < b ? a : b)
#define MAX(a, b) (a > b ? a : b)

#define EPS 1e-7

// gpu defined properties
#define GRID_SIZE  32
#define BLOCK_SIZE 32

// some hacks
#define LOG_NUM_BANKS 5 // for 32
#define CONFLICT_FREE_OFFSET(i) ((i) >> LOG_NUM_BANKS)


// sort properties
#define BUCKET_SIZE 1024
#define SPLIT_SIZE  512


#define INDEX_FROM_FLOAT_VALUE(value,min,max,count) (int)((value-min)/(max-min)*(count-1))
#define SWAP_FLOATS(a,b) {float t = a; a = b; b = t;}





// recursion depth control
int depth = 0;
int max_depth = 0;

void depth_inc() {
    depth++;
    if (depth > max_depth) {
        max_depth = depth;
    }
}

void depth_dec() {
    depth--;
}

void print_depth_bar() {
    for (int i = 0; i < depth; i++) {
        printf("__");
    }
}

void print_depth_space() {
    for (int i = 0; i < depth; i++) {
        printf("  ");
    }
}



// =============================================================================
//                                   PRINT
// =============================================================================

void print_array(float *data, int size) {
    for (int i = 0; i < size; i++) {
        printf("%f ", data[i]);
    }
    printf("\n");
}

void print_array(int *data, int size) {
    for (int i = 0; i < size; i++) {
        printf("%d ", data[i]);
    }
    printf("\n");
}



// =============================================================================
//                              DATA READ/WRITE
// =============================================================================

float *read_data(int *n) {
    fread(n, sizeof(int), 1, stdin);
    float *data = (float *)malloc(*n * sizeof(float));
    fread(data, sizeof(float), *n, stdin);
    return data;
}

float *read_data_as_plain_text(int *n) {
    scanf("%d", n);
    float *data = (float *)malloc(*n * sizeof(float));
    int count = 0;
    for (int i = 0; i < *n; i++) {
        scanf("%f", &data[i]);
        count++;
    }
#ifdef DEBUG
    printf("data count: %d\n", count);
#endif

    return data;
}

void write_data(float *data, int n) {
    fwrite(data, sizeof(float), n, stdout);
}

void write_data_with_size(float *data, int n) {
    fwrite(&n, sizeof(int), 1, stdout);
    fwrite(data, sizeof(float), n, stdout);
}





// =============================================================================
//                                   REDUCE
// =============================================================================

__global__ void gpuReduceMaxFloat(float *data, int n, float *result) {

    __shared__ float shared_data[2048];

    int global_thread_id = 2 * blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id = threadIdx.x;

    if (global_thread_id + blockDim.x < n) {
        shared_data[thread_id] = MAX(data[global_thread_id], data[global_thread_id + blockDim.x]);
    } else if (global_thread_id < n) {
        shared_data[thread_id] = data[global_thread_id];
    } else {
        shared_data[thread_id] = data[0]; // just dummy
    }

    __syncthreads();;

    // reduction process
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (thread_id < i) {
            shared_data[thread_id] = MAX(shared_data[thread_id], shared_data[thread_id + i]);
        }
        __syncthreads();
    }

    // write result to global memory
    if (thread_id == 0) {
        result[blockIdx.x] = shared_data[0];
    }
}

__global__ void gpuReduceMinFloat(float *data, int n, float *result) {

    __shared__ float shared_data[2048];

    int global_thread_id = 2 * blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id = threadIdx.x;

    if (global_thread_id + blockDim.x < n) {
        shared_data[thread_id] = MIN(data[global_thread_id], data[global_thread_id + blockDim.x]);
    } else if (global_thread_id < n) {
        shared_data[thread_id] = data[global_thread_id];
    } else {
        shared_data[thread_id] = data[0]; // just dummy
    }

    __syncthreads();;

    // reduction process
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (thread_id < i) {
            shared_data[thread_id] = MIN(shared_data[thread_id], shared_data[thread_id + i]);
        }
        __syncthreads();
    }

    // write result to global memory
    if (thread_id == 0) {
        result[blockIdx.x] = shared_data[0];
    }
}

__host__ void recursive_gpu_reduce_max(float *data_device, int n, float *result_host) {
    int gridSize = (n / 2048) + 1;
    int blockSize = 1024;

    // printf("gridSize = %d\n", gridSize);

    float *result_device = NULL;
    CSC(hipMalloc((void **)&result_device, gridSize * sizeof(float)));
    CSC(hipGetLastError());

    if (result_device == NULL) {
        printf("YOLO! NULL MEMORY DETECTED!\n");
    }

    gpuReduceMaxFloat <<<gridSize, blockSize>>> (data_device, n, result_device);
    CSC(hipGetLastError());

    if (gridSize > 1) {
        recursive_gpu_reduce_max(result_device, gridSize, result_host);
    } else {
        CSC(hipMemcpy(result_host, result_device, sizeof(float), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());
    }
    CSC(hipFree(result_device));
    CSC(hipGetLastError());
}

__host__ void recursive_gpu_reduce_min(float *data_device, int n, float *result_host) {
    int gridSize = (n / 2048) + 1;
    int blockSize = 1024;

    // printf("gridSize = %d\n", gridSize);

    float *result_device = NULL;
    CSC(hipMalloc((void **)&result_device, gridSize * sizeof(float)));
    CSC(hipGetLastError());

    if (result_device == NULL) {
        print_depth_space();
        printf("YOLO! NULL MEMORY DETECTED!\n");
    }

#ifdef DEBUG
    print_depth_space();
    printf("before reduce min: n = %d, gridSize = %d, blockSize = %d\n", n, gridSize, blockSize);
#endif

    gpuReduceMinFloat <<<gridSize, blockSize>>> (data_device, n, result_device);
    CSC(hipGetLastError());

    if (gridSize > 1) {
        recursive_gpu_reduce_min(result_device, gridSize, result_host);
    } else {
        CSC(hipMemcpy(result_host, result_device, sizeof(float), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());
    }
    CSC(hipFree(result_device));
    CSC(hipGetLastError());
}






// =============================================================================
//                                    SCAN
// =============================================================================

__global__ void scan3(int *data, int n, int *sums, int *result) {
    // __shared__ int temp[2 * BLOCK_SIZE + CONFLICT_FREE_OFFSET(2 * BLOCK_SIZE)];

    __shared__ int temp[2 * 512 + CONFLICT_FREE_OFFSET(2 * 512)];

    int thread_id = threadIdx.x;
    int offset = 1;
    int ai = thread_id;
    // int bi = thread_id + (n / 2);  // different with Roma's code
    int bi = thread_id + 512;
    int offset_A = CONFLICT_FREE_OFFSET(ai);
    int offset_B = CONFLICT_FREE_OFFSET(bi);

    // printf("scan3 [ai + offset_A] = [%d]\nscan3 [bi + offset_B] = [%d]\n", ai + offset_A, bi + offset_B);

    temp[ai] = 0;
    temp[bi] = 0;

    __syncthreads();

    if (ai + 2 * 512 * blockIdx.x < n) {
        temp[ai + offset_A] = data[ai + 2 * 512 * blockIdx.x];
    } else {
        temp[ai + offset_A] = 0;
    }

    if (bi + 2 * 512 * blockIdx.x < n) {
        temp[ai + offset_B] = data[bi * 2 * 512 * blockIdx.x];
    } else {
        temp[bi + offset_B] = 0;
    }

    // temp[ai + offset_A] = data[ai + 2 * BLOCK_SIZE * blockIdx.x];
    // temp[bi + offset_B] = data[bi + 2 * BLOCK_SIZE * blockIdx.x];

    // for (int d = n >> 1; d > 0; d >>= 1) {
    for (int d = 512; d > 0; d >>= 1) {
        __syncthreads();
        if (thread_id < d) {
            int ai = offset * (2 * thread_id + 1) - 1;
            int bi = offset * (2 * thread_id + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            temp[bi] += temp[ai];
        }
        offset <<= 1;
    }

    if (thread_id == 0) {
        // int idx = n - 1 + CONFLICT_FREE_OFFSET(n - 1);
        int idx = 2 * 512 - 1 + CONFLICT_FREE_OFFSET(2 * 512 - 1);
        sums[blockIdx.x] = temp[idx];
        temp[idx] = 0;
    }

    __syncthreads();

    // for (int d = 1; d < n; d <<= 1) {
    for (int d = 1; d < 2 * 512; d <<= 1) {
        offset >>= 1;
        __syncthreads();
        if (thread_id < d) {
            int ai = offset * (2 * thread_id + 1) - 1;
            int bi = offset * (2 * thread_id + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }

    __syncthreads();

    // offset = 2 * BLOCK_SIZE * blockIdx.x;
    offset = 2 * 512 * blockIdx.x;

    // printf("scan3 indexes = (%d, %d)\n", ai + offset, bi + offset);

    if (ai + offset < n) {
        result[ai + offset] = temp[ai + offset];
    }

    if (bi + offset < n) {
        result[bi + offset] = temp[bi + offset];
    }

    // result[ai + offset] = temp[ai + offset_A];
    // result[bi + offset] = temp[bi + offset_B];
}

__global__ void scanDistribute(int *data, int n, int *sums) {
    //int idx = threadIdx.x + blockIdx.x * 2 * BLOCK_SIZE;
    int idx = threadIdx.x + blockIdx.x * 2 * 512;
    if (idx < n) {
        data[idx] += sums[blockIdx.x];
    }
    // printf("scanDistribute index = %d\n", idx);
    // data[idx] += sums[blockIdx.x];
}

__host__ void recursive_gpu_scan(int *data, int n, int *result) {

#ifdef DEBUG
    print_depth_space();
    printf("recursive_gpu_scan (data size = %d)\n", n);
#endif

    // int numBlocks = n / (2 * BLOCK_SIZE) + 1;

    int threadsPerBlock = 512;
    int threads = 512 * 2;
    int numBlocks = n / (512 * 2) + 1;

#ifdef DEBUG
    print_depth_space();
    printf("numBlocks = %d\n", numBlocks);
#endif

    int *sums  = NULL;
    int *sums2 = NULL;

    CSC(hipMalloc((void **)&sums, numBlocks * sizeof(int)));
    CSC(hipMemset(sums, 0, numBlocks * sizeof(int)));
    CSC(hipGetLastError());

    CSC(hipMalloc((void **)&sums2, numBlocks * sizeof(int)));
    CSC(hipMemset(sums2, 0, numBlocks * sizeof(int)));
    CSC(hipGetLastError());

    // dim3 threads(BLOCK_SIZE, 1, 1);
    // dim3 blocks(numBlocks, 1, 1);

    // scan3 <<<blocks, threads>>> (data, 2 * BLOCK_SIZE, sums, result);
    scan3 <<<numBlocks, threadsPerBlock>>> (data, n, sums, result);
    CSC(hipGetLastError());

    // if (n >= 2 * BLOCK_SIZE) {
    if (n >= threads) {
        // printf("%d >= 2 * %d\n", n, BLOCK_SIZE);
        recursive_gpu_scan(sums, numBlocks, sums2);
        CSC(hipGetLastError());
    } else {
        CSC(hipMemcpy(sums2, sums, numBlocks * sizeof(int), hipMemcpyDeviceToDevice));
        CSC(hipGetLastError());
    }

    // if (numBlocks > 1) {
    if (numBlocks - 1 > 0) {
        // threads = dim3(2 * BLOCK_SIZE, 1, 1);
        // blocks = dim3(numBlocks - 1, 1, 1);

        dim3 blocks(numBlocks - 1, 1, 1);
        dim3 threads(1024, 1, 1);

#ifdef DEBUG
        print_depth_space();
        printf("before distribute: blocks = %d, threads = %d\n", blocks.x, threads.x);
#endif

        // scanDistribute <<<blocks, threads>>> (result + (2 * BLOCK_SIZE), sums2 + 1);
        scanDistribute <<<blocks, threads>>> (result + 1024, n - 1024, sums2 + 1);

        CSC(hipGetLastError());
    }

    hipFree(sums);
    CSC(hipGetLastError());

    hipFree(sums2);
    CSC(hipGetLastError());
}


// =============================================================================
//                                  HISTOGRAM
// =============================================================================

// gpu histogram
__global__ void gpuHistogramCalculateSplitsSizes(float *data, int n, int *result, float min, float max, int splits_count) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    for ( ; index < n; index += offset) {
        int insertion_index = INDEX_FROM_FLOAT_VALUE(data[index], min, max, splits_count);
        atomicAdd(&(result[insertion_index]), 1);
    }
}

__global__ void gpuHistogramFillSplits(float *data_device, int n, float *splits_device,
                                       int *begin_position_for_split_device,
                                       unsigned int *current_size_of_split_device,
                                       float min, float max, int splits_count)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    for ( ; index < n; index += offset) {
        float value = data_device[index];
        int split_index = INDEX_FROM_FLOAT_VALUE(value, min, max, splits_count);
        // int current_size_of_split = atomicInc(&(current_size_of_split_device[split_index]), 1); // WARNING! POTENTIAL ERROR!
        int current_size_of_split = atomicAdd(&(current_size_of_split_device[split_index]), 1); /// ?????
        int insert_position = begin_position_for_split_device[split_index] + current_size_of_split;
        splits_device[insert_position] = value;
#ifdef DEBUG
        // printf("Block_id = %d, thread_id = %d -- insert value [%f] from index [%d] to splits index [%d]\n", blockIdx.x, threadIdx.x, value, index, insert_position);
#endif
    }
}



// =============================================================================
//                                   SORT
// =============================================================================

void swap(float *lhs, float *rhs) {
    float temp = *lhs;
    *lhs = *rhs;
    *rhs = temp;
}


void odd_even_sort(float *data, int size) {
    for (int i = 0; i < size; i++) {
        for (int j = i & 1; j < size - 1; j += 2) {
            if (data[j] > data[j + 1]) {
                swap(&data[j], &data[j + 1]);
            }
        }
    }
}

// multiple threads
__global__ void oddEvenSort(float *data, int n, int buckets_count, int *begin_position_for_bucket, int *size_of_bucket) {
    int bucket_index = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;

    for (int b = bucket_index; b < buckets_count; b += offset) {
        int size = size_of_bucket[bucket_index];
        if (size == -1) { // already sorted
            continue;
        }
        int begin = begin_position_for_bucket[b];
        for (int i = begin; i < begin + size; i++) {
            for (int j = i & 1; j < begin + size - 1; j += 2) {
                if (data[j] > data[j + 1]) {
                    float temp = data[j];
                    data[j] = data[j + 1];
                    data[j + 1] = temp;
                }
            }
        }
    }
}

// 1 block - 1 bucket
__global__ void gpuOddEvenSort(float *buckets, int n, int *begin_position_for_bucket, int *size_of_bucket) {

    int bucket_index = blockIdx.x;
    int bucket_size = size_of_bucket[bucket_index];

    if (bucket_size == -1) { // bucket already sorted
        return;
    }

    // prepare shared array for bucket
    __shared__ float shared_bucket[BUCKET_SIZE];

    int thread_id = threadIdx.x;

    shared_bucket[2 * thread_id    ] = FLT_MAX; // dummy for item index out of bounds
    shared_bucket[2 * thread_id + 1] = FLT_MAX; // dummy for item index out of bounds

    __syncthreads();

    // load bucket items
    int item_index = 0;

    item_index = 2 * thread_id + begin_position_for_bucket[bucket_index];
    if (item_index - begin_position_for_bucket[bucket_index] < bucket_size) {
        shared_bucket[2 * thread_id] = buckets[item_index];
    }

    item_index = 2 * thread_id + 1 + begin_position_for_bucket[bucket_index];
    if (item_index - begin_position_for_bucket[bucket_index] < bucket_size) {
        shared_bucket[2 * thread_id + 1] = buckets[item_index];
    }

    __syncthreads();

    int  odd_index_limit = BUCKET_SIZE - 1;
    int even_index_limit = BUCKET_SIZE;

    for (int i = 0; i < blockDim.x; i++) {
        item_index = 2 * thread_id + 1;
        if (item_index < odd_index_limit) { // is it correct ??
            if (shared_bucket[item_index] > shared_bucket[item_index + 1]) {
                SWAP_FLOATS(shared_bucket[item_index], shared_bucket[item_index + 1]);
            }
        }
        __syncthreads();
        item_index = 2 * thread_id;
        if (thread_id < even_index_limit) { // is it correct ??
            if (shared_bucket[item_index] > shared_bucket[item_index + 1]) {
                SWAP_FLOATS(shared_bucket[item_index], shared_bucket[item_index + 1]);
            }
        }
        __syncthreads();
    }

    // write result back

    item_index = 2 * thread_id + begin_position_for_bucket[bucket_index];
    if (item_index - begin_position_for_bucket[bucket_index] < bucket_size) {
        buckets[item_index] = shared_bucket[2 * thread_id];

    }

    item_index = 2 * thread_id + 1 + begin_position_for_bucket[bucket_index];
    if (item_index - begin_position_for_bucket[bucket_index] < bucket_size) {
        buckets[item_index] = shared_bucket[2 * thread_id + 1];
    }

    __syncthreads(); // why?
}




//
// gpu_bucket_sort description:
//  data_device -- initial array allocated for GPU usage;
//            n -- amount of items in data array;
//
__host__ void gpu_bucket_sort(float *data_device, int n) {

#ifdef DEBUG
    depth_inc();
    print_depth_space();
    printf("BEGIN SORT\n");
#endif

    // find min data value
    float min = FLT_MAX;
    recursive_gpu_reduce_min(data_device, n, &min);
    CSC(hipGetLastError());

    // find max data value
    float max = -FLT_MAX;
    recursive_gpu_reduce_max(data_device, n, &max);
    CSC(hipGetLastError());


#ifdef DEBUG
    print_depth_space();
    printf("MIN = %f, MAX = %f\n", min, max);
#endif

    // check for already sorted array
    if (fabs(min - max) < EPS) {
#ifdef DEBUG
        depth_dec();
#endif
        return;
    }

    // make splits
    int splits_count = n / SPLIT_SIZE + 1;

#ifdef DEBUG
    print_depth_space();
    printf("splits_count = %d\n", splits_count);
#endif

    // create size_of_split on gpu
    int *size_of_split_device = NULL;
    CSC(hipMalloc((void **)&size_of_split_device, splits_count * sizeof(int)));
    CSC(hipMemset(size_of_split_device, 0, splits_count * sizeof(int)));
    CSC(hipGetLastError());

    // calculate splits sizes with histogram
    gpuHistogramCalculateSplitsSizes <<<GRID_SIZE, BLOCK_SIZE>>> (data_device, n, size_of_split_device, min, max, splits_count);
    CSC(hipGetLastError());


#ifdef DEBUG // check size_of_split array

    // int *size_of_split = (int *)malloc(splits_count * sizeof(int));
    // memset(size_of_split, 0, splits_count * sizeof(int));
    //
    // CSC(cudaMemcpy(size_of_split, size_of_split_device, splits_count * sizeof(int), cudaMemcpyDeviceToHost));
    // CSC(cudaGetLastError());
    //
    // print_depth_space();
    // printf("size_of_split: ");
    // print_array(size_of_split, splits_count);
    //
    // free(size_of_split);

#endif

    // calculate splits begin position with scan
    int *begin_position_for_split_device = NULL;
    CSC(hipMalloc((void **)&begin_position_for_split_device, splits_count * sizeof(int)));
    CSC(hipGetLastError());

    recursive_gpu_scan(size_of_split_device, n, begin_position_for_split_device);
    CSC(hipGetLastError());


#ifdef DEBUG // check begin_position_for_split

    // int *begin_position_for_split = (int *)malloc(splits_count * sizeof(int));
    // CSC(cudaMemcpy(begin_position_for_split, begin_position_for_split_device, splits_count * sizeof(int), cudaMemcpyDeviceToHost));
    // CSC(cudaGetLastError());
    //
    // print_depth_space();
    // printf("begin_position_for_split: ");
    // print_array(begin_position_for_split, splits_count);
    //
    // free(begin_position_for_split);

#endif



    unsigned int *current_size_of_split_device = NULL;
    CSC(hipMalloc((void **)&current_size_of_split_device, splits_count * sizeof(unsigned int)));
    CSC(hipGetLastError());

    CSC(hipMemset(current_size_of_split_device, 0, splits_count * sizeof(unsigned int)));
    CSC(hipGetLastError());

    // create splits array
    float *splits_device = NULL;
    CSC(hipMalloc((void **)&splits_device, n * sizeof(float)));
    CSC(hipGetLastError());

    // fill splits with histogram
    gpuHistogramFillSplits <<<GRID_SIZE, BLOCK_SIZE>>> (data_device, n, splits_device,
                                                        begin_position_for_split_device,
                                                        current_size_of_split_device,
                                                        min, max, splits_count);
    CSC(hipGetLastError());


#ifdef DEBUG // check splits array

    // float *splits = (float *)malloc(n * sizeof(float));
    // CSC(cudaMemcpy(splits, splits_device, n * sizeof(float), cudaMemcpyDeviceToHost));
    // CSC(cudaGetLastError());
    //
    // print_depth_space();
    // printf("splits: ");
    // print_array(splits, n);
    //
    // free(splits);

#endif




    // make buckets
    int buckets_count = splits_count;
    int *size_of_bucket = (int *)malloc(buckets_count * sizeof(int));
    memset(size_of_bucket, 0, buckets_count * sizeof(int));

    int *begin_position_for_bucket = (int *)malloc(buckets_count * sizeof(int));

    int bucket_index = 0;

    for (int split_index = 0; split_index < splits_count; split_index++) {

        int split_size = 0;
        CSC(hipMemcpy(&split_size, &(size_of_split_device[split_index]), sizeof(int), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());

        int split_begin_position = 0;
        CSC(hipMemcpy(&split_begin_position, &(begin_position_for_split_device[split_index]), sizeof(int), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());

#ifdef DEBUG
        // print_depth_space();
        // printf("split_size = %d\n", split_size);
        // print_depth_space();
        // printf("split_begin_position = %d\n", split_begin_position);
#endif

        if (split_size > BUCKET_SIZE) {

            bucket_index++;

            // sort current split
            float *split = &(splits_device[split_begin_position]); // gpu pointer
            gpu_bucket_sort(split, split_size);

            // remember split as bucket
            begin_position_for_bucket[bucket_index] = split_begin_position; // ????
            // size_of_bucket[bucket_index] = split_size; // ????
            size_of_bucket[bucket_index] = -1; // -1 indicates that bucket already sorted
            bucket_index++;

        } else {

            int current_bucket_remaining_capacity = BUCKET_SIZE - size_of_bucket[bucket_index];
            if (split_size <= current_bucket_remaining_capacity) {
                // insert split to current bucket
                if (current_bucket_remaining_capacity == BUCKET_SIZE) {
                    begin_position_for_bucket[bucket_index] = split_begin_position;
                }
                size_of_bucket[bucket_index] += split_size;

            } else {
                // insert split to next bucket
                bucket_index++;
                begin_position_for_bucket[bucket_index] = split_begin_position;
                size_of_bucket[bucket_index] = split_size;
            }
        }
    }

    // determine correct buckets count
    if (size_of_bucket[bucket_index] == 0) {
        buckets_count = bucket_index;
    } else {
        buckets_count = bucket_index + 1;
    }

    // sort buckets

    int *begin_position_for_bucket_device = NULL;
    CSC(hipMalloc((void **)&begin_position_for_bucket_device, buckets_count * sizeof(int)));
    CSC(hipGetLastError());
    CSC(hipMemcpy(begin_position_for_bucket_device, begin_position_for_bucket, buckets_count * sizeof(int), hipMemcpyHostToDevice));
    CSC(hipGetLastError());

    int *size_of_bucket_device = NULL;
    CSC(hipMalloc((void **)&size_of_bucket_device, buckets_count * sizeof(int)));
    CSC(hipGetLastError());
    CSC(hipMemcpy(size_of_bucket_device, size_of_bucket, buckets_count * sizeof(int), hipMemcpyHostToDevice));
    CSC(hipGetLastError());

    // oddEvenSort <<<1024, 1024>>> (splits_device, n, buckets_count, begin_position_for_bucket_device, size_of_bucket_device);

    dim3 sortBlocks(buckets_count, 1, 1);
    dim3 sortThreads(BUCKET_SIZE, 1, 1);

    gpuOddEvenSort <<<sortBlocks, sortThreads>>> (splits_device, n, begin_position_for_bucket_device, size_of_bucket_device);
    CSC(hipGetLastError());

    // for (int i = 0; i < buckets_count; i++) {
    //     int bucket_size = size_of_bucket[i];
    //     if (bucket_size == -1) { // already sorted
    //         continue;
    //     }
    //     float *bucket = (float *)malloc(bucket_size * sizeof(float));
    //     int bucket_begin_position = begin_position_for_bucket[i];
    //
    //     CSC(cudaMemcpy(bucket, &(splits_device[bucket_begin_position]), bucket_size * sizeof(float), cudaMemcpyDeviceToHost));
    //     CSC(cudaGetLastError());
    //
    //     odd_even_sort(bucket, bucket_size);
    //
    //     CSC(cudaMemcpy(&(splits_device[bucket_begin_position]), bucket, bucket_size * sizeof(float), cudaMemcpyHostToDevice));
    //     CSC(cudaGetLastError());
    //
    //     free(bucket);
    // }

    CSC(hipMemcpy(data_device, splits_device, n * sizeof(float), hipMemcpyDeviceToDevice));

    CSC(hipFree(size_of_split_device));
    CSC(hipFree(splits_device));
    CSC(hipFree(begin_position_for_split_device));
    CSC(hipFree(current_size_of_split_device));
    CSC(hipFree(begin_position_for_bucket_device));
    CSC(hipFree(size_of_bucket_device));
    CSC(hipGetLastError());

    free(size_of_bucket);
    free(begin_position_for_bucket);

#ifdef DEBUG
    print_depth_space();
    printf("END SORT\n");
    depth_dec();
#endif

}


//
// bucket_sort description:
//  data -- initial array allocated for CPU usage;
//     n -- amount of items in data array;
//
__host__ void bucket_sort(float *data, int n) {
    // prepare device data
    float *data_device = NULL;
    CSC(hipMalloc((void **)&data_device, n * sizeof(float)));
    CSC(hipMemcpy(data_device, data, n * sizeof(float), hipMemcpyHostToDevice));
    CSC(hipGetLastError());

    gpu_bucket_sort(data_device, n);

    CSC(hipMemcpy(data, data_device, n * sizeof(float), hipMemcpyDeviceToHost));
    CSC(hipGetLastError());
}


// =============================================================================
//                                  MAIN
// =============================================================================

bool sorted(float *data, int n) {
    for (int i = 1; i < n; i++) {
        if (data[i] < data[i - 1]) {
            return false;
        }
    }
    return true;
}


int main() {

    int n = 0;
    float *data = read_data_as_plain_text(&n);
    // float *data = read_data(&n);

    if (n == 0) {
        free(data);
        return 0;
    }

    bucket_sort(data, n);

    // print_array(data, n);

    if (sorted(data, n)) {
        printf("--\nStatus: OK\n");
    } else {
        printf("--\nStatus: WA\n");
    }

    free(data);

    return 0;
}
