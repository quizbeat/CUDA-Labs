#include "hip/hip_runtime.h"
//
//  lab3.cu
//  CUDA-Lab-3
//
//  Created by Nikita Makarov on 07/05/16.
//  Copyright © 2016 Nikita Makarov. All rights reserved.
//

#include <iostream>
#include <stdio.h>
#include <limits>
#include <cfloat>
#include <math.h>
#include <time.h>
#include <fstream>

#define CSC(call) {														\
    hipError_t err = call;												\
    if(err != hipSuccess) {											\
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	\
            __FILE__, __LINE__, hipGetErrorString(err));				\
        exit(1);														\
    }																	\
} while (0)


#define MIN(a, b) (a < b ? a : b)
#define MAX(a, b) (a > b ? a : b)
#define EPS 1e-7
#define BUCKET_SIZE 1024
#define SPLIT_SIZE  512




// recursion depth control
int depth = 0;
int max_depth = 0;

void depth_inc() {
    depth++;
    if (depth > max_depth) {
        max_depth = depth;
    }
}

void depth_dec() {
    depth--;
}

void print_depth_bar() {
    for (int i = 0; i < depth; i++) {
        printf("__");
    }
}




// =============================================================================
//                                   PRINT
// =============================================================================

void print_array(float *data, int size) {
    for (int i = 0; i < size; i++) {
        printf("%f ", data[i]);
    }
    printf("\n");
}

void print_subarray(float *data, int begin, int end) {
    for (int i = begin; i < end; i++) {
        printf("%f ", data[i]);
    }
    printf("\n");
}

void print_array(int *data, int size) {
    for (int i = 0; i < size; i++) {
        printf("%d ", data[i]);
    }
    printf("\n");
}

void print_int_address(int *p) {
    printf(">>>> INT   Address = %p,  Value = %d\n", p, *p);
}

void print_float_address(float *p) {
    printf(">>>> FLOAT Address = %p,  Value = %f\n", p, *p);
}




// =============================================================================
//                              DATA READ/WRITE
// =============================================================================

float *read_data(int *n) {
    fread(n, sizeof(int), 1, stdin);
    float *data = (float *)malloc(*n * sizeof(float));
    fread(data, sizeof(float), *n, stdin);
    return data;
}

float *read_data_as_plain_text(int *n) {
    scanf("%d", n);
    float *data = (float *)malloc(*n * sizeof(float));
    int count = 0;
    for (int i = 0; i < *n; i++) {
        scanf("%f", &data[i]);
        count++;
    }
    printf("data count: %d\n", count);
    return data;
}

void write_data(float *data, int n) {
    fwrite(data, sizeof(float), n, stdout);
}

void write_data_with_size(float *data, int n) {
    fwrite(&n, sizeof(int), 1, stdout);
    fwrite(data, sizeof(float), n, stdout);
}




// =============================================================================
//                                    HELPERS
// =============================================================================

float max_float(float lhs, float rhs) {
    return (lhs > rhs) ? lhs : rhs;
}

float min_float(float lhs, float rhs) {
    return (lhs < rhs) ? lhs : rhs;
}

float sum_float(float lhs, float rhs) {
    return lhs + rhs;
}

float multiply_float(float lhs, float rhs) {
    return lhs * rhs;
}

int sum_int(int lhs, int rhs) {
    return lhs + rhs;
}

// map float value to split index
int index_from_float_value(float value, float min, float max, int splits_count) {
    int index = (int)((value - min) / (max - min) * (splits_count - 1));
    return index;
}

void swap(float *lhs, float *rhs) {
    float temp = *lhs;
    *lhs = *rhs;
    *rhs = temp;
}



// =============================================================================
//                                   REDUCE
// =============================================================================

// plain cpu reduce
void reduce(float *data, int n, float *result, float (*op)(float, float), float identity) {
    *result = identity;
    for (int i = 0; i < n; i++) {
        *result = op(data[i], *result);
    }
}

void reduce_min_max(float *data, int size, float *min, float *max) {
    *min =  FLT_MAX;
    *max = -FLT_MAX;
    for (int i = 0; i < size; i++) {
        if (data[i] > *max) { *max = data[i]; }
        if (data[i] < *min) { *min = data[i]; }
    }
}



// gpu reduce
__global__ void gpuReduceSumFloat(float *data, int n, float *result) {

    __shared__ float shared_data[2048];

    int global_thread_id = 2 * blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id = threadIdx.x;

    if (global_thread_id + blockDim.x < n) {
        shared_data[thread_id] = data[global_thread_id] + data[global_thread_id + blockDim.x];
    } else if (global_thread_id < n) {
        shared_data[thread_id] = data[global_thread_id];
    } else {
        shared_data[thread_id] = 0;
    }

    __syncthreads();;

    // reduction process
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (thread_id < i) {
            shared_data[thread_id] += shared_data[thread_id + i];
        }
        __syncthreads();
    }

    // write result to global memory
    if (thread_id == 0) {
        result[blockIdx.x] = shared_data[0];
    }
}

__global__ void gpuReduceMaxFloat(float *data, int n, float *result) {

    __shared__ float shared_data[2048];

    int global_thread_id = 2 * blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id = threadIdx.x;

    if (global_thread_id + blockDim.x < n) {
        shared_data[thread_id] = MAX(data[global_thread_id], data[global_thread_id + blockDim.x]);
    } else if (global_thread_id < n) {
        shared_data[thread_id] = data[global_thread_id];
    } else {
        shared_data[thread_id] = data[0]; // just dummy
    }

    __syncthreads();;

    // reduction process
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (thread_id < i) {
            shared_data[thread_id] = MAX(shared_data[thread_id], shared_data[thread_id + i]);
        }
        __syncthreads();
    }

    // write result to global memory
    if (thread_id == 0) {
        result[blockIdx.x] = shared_data[0];
    }
}

__global__ void gpuReduceMinFloat(float *data, int n, float *result) {

    __shared__ float shared_data[2048];

    int global_thread_id = 2 * blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id = threadIdx.x;

    if (global_thread_id + blockDim.x < n) {
        shared_data[thread_id] = MIN(data[global_thread_id], data[global_thread_id + blockDim.x]);
    } else if (global_thread_id < n) {
        shared_data[thread_id] = data[global_thread_id];
    } else {
        shared_data[thread_id] = data[0]; // just dummy
    }

    __syncthreads();;

    // reduction process
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (thread_id < i) {
            shared_data[thread_id] = MIN(shared_data[thread_id], shared_data[thread_id + i]);
        }
        __syncthreads();
    }

    // write result to global memory
    if (thread_id == 0) {
        result[blockIdx.x] = shared_data[0];
    }
}

__host__ void recursive_gpu_reduce_sum(float *data_device, int n, float *result_host) {
    int gridSize = (n / 2048) + 1;
    int blockSize = 1024;

    // printf("gridSize = %d\n", gridSize);

    float *result_device = NULL;
    CSC(hipMalloc((void **)&result_device, gridSize * sizeof(float)));
    CSC(hipGetLastError());

    if (result_device == NULL) {
        printf("YOLO! NULL MEMORY DETECTED!\n");
    }

    gpuReduceSumFloat <<<gridSize, blockSize>>> (data_device, n, result_device);
    CSC(hipGetLastError());

    if (gridSize > 1) {
        recursive_gpu_reduce_sum(result_device, gridSize, result_host);
    } else {
        CSC(hipMemcpy(result_host, result_device, sizeof(float), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());
    }
    CSC(hipFree(result_device));
    CSC(hipGetLastError());
}

__host__ void recursive_gpu_reduce_max(float *data_device, int n, float *result_host) {
    int gridSize = (n / 2048) + 1;
    int blockSize = 1024;

    // printf("gridSize = %d\n", gridSize);

    float *result_device = NULL;
    CSC(hipMalloc((void **)&result_device, gridSize * sizeof(float)));
    CSC(hipGetLastError());

    if (result_device == NULL) {
        printf("YOLO! NULL MEMORY DETECTED!\n");
    }

    gpuReduceMaxFloat <<<gridSize, blockSize>>> (data_device, n, result_device);
    CSC(hipGetLastError());

    if (gridSize > 1) {
        recursive_gpu_reduce_max(result_device, gridSize, result_host);
    } else {
        CSC(hipMemcpy(result_host, result_device, sizeof(float), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());
    }
    CSC(hipFree(result_device));
    CSC(hipGetLastError());
}

__host__ void recursive_gpu_reduce_min(float *data_device, int n, float *result_host) {
    int gridSize = (n / 2048) + 1;
    int blockSize = 1024;

    // printf("gridSize = %d\n", gridSize);

    float *result_device = NULL;
    CSC(hipMalloc((void **)&result_device, gridSize * sizeof(float)));
    CSC(hipGetLastError());

    if (result_device == NULL) {
        printf("YOLO! NULL MEMORY DETECTED!\n");
    }

    gpuReduceMinFloat <<<gridSize, blockSize>>> (data_device, n, result_device);
    CSC(hipGetLastError());

    if (gridSize > 1) {
        recursive_gpu_reduce_min(result_device, gridSize, result_host);
    } else {
        CSC(hipMemcpy(result_host, result_device, sizeof(float), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());
    }
    CSC(hipFree(result_device));
    CSC(hipGetLastError());
}






// =============================================================================
//                                    SCAN
// =============================================================================

// plain cpu scan
// floating numbers scan
void scan(float *data, int n, float *result_data, float (*op)(float, float), float identity, bool inclusive) {
    float result = identity;
    for (int i = 0; i < n; i++) {
        if (inclusive) {
            result = op(result, data[i]);
            result_data[i] = result;
        } else { // exclusive
            result_data[i] = result;
            result = op(result, data[i]);
        }
    }
}

// integer numbers scan
void scan(int *data, int n, int *result_data, int (*op)(int, int), int identity, bool inclusive) {
    int result = identity;
    for (int i = 0; i < n; i++) {
        if (inclusive) {
            result = op(result, data[i]);
            result_data[i] = result;
        } else { // exclusive
            result_data[i] = result;
            result = op(result, data[i]);
        }
    }
}

void scan_calculate_positions(int *position_for_split, int n, int *size_of_split) {
    for (int i = 1; i < n; i++) {
        position_for_split[i] = position_for_split[i - 1] + size_of_split[i - 1];
    }
}




// =============================================================================
//                                  HISTOGRAM
// =============================================================================

void histogram(float *data, int n, int *result, int op(float value, float min, float max, int count), float min, float max, int count) {
    for (int i = 0; i < n; i++) {
        int index = op(data[i], min, max, count);
        result[index]++;
    }
}

void histogram_splits_sizes(float *data, int n, int *size_of_split, float min, float max, int splits_count) {
    for (int i = 0; i < n; i++) {
        int index = index_from_float_value(data[i], min, max, splits_count);
        size_of_split[index]++;
    }
}

void histogram_fill_splits(float *data, int n, float *splits, int *begin_postion_for_split, int *current_size_of_split, float min, float max, int splits_count) {
    for (int i = 0; i < n; i++) {
        float value = data[i];
        int split_index = index_from_float_value(value, min, max, splits_count);
        int insert_position = begin_postion_for_split[split_index] + current_size_of_split[split_index];
        splits[insert_position] = value;
        current_size_of_split[split_index]++;
    }
}




// =============================================================================
//                                   SORT
// =============================================================================

void odd_even_sort(float *data, int size) {
    for (int i = 0; i < size; i++) {
        for (int j = i & 1; j < size - 1; j += 2) {
            if (data[j] > data[j + 1]) {
                swap(&data[j], &data[j + 1]);
            }
        }
    }
}

void odd_even_sort_range(float *data, int begin, int end) {
    int size = end - begin;
    for (int i = 0; i < size; i++) {
        for (int j = i & 1; j < size - 1; j += 2) {
            if (data[j + begin] > data[j + begin + 1]) {
                swap(&data[j + begin], &data[j + begin + 1]);
            }
        }
    }
}

void bucket_sort(float *data, int n) {

    depth_inc();

    float min = FLT_MAX;
    float max = -FLT_MAX;

    reduce(data, n, &min, &min_float,  FLT_MAX);
    reduce(data, n, &max, &max_float, -FLT_MAX);

    if (fabs(min - max) < EPS) { // data already sorted
        depth_dec();
        return;
    }


    // STEP 2: Make splits

    // calculate splits count
    int splits_count = n / SPLIT_SIZE + 1;


    // create array for splits sizes
    int *size_of_split = (int *)malloc(splits_count * sizeof(int));
    for (int i = 0; i < splits_count; i++) {
        size_of_split[i] = 0;
    }


    // perform histogram to calculate splits sizes
    histogram(data, n, size_of_split, &index_from_float_value, min, max, splits_count);


    // calculate splits begin position
    int *begin_position_for_split = (int *)malloc(splits_count * sizeof(int));
    scan(size_of_split, splits_count, begin_position_for_split, &sum_int, 0, false);


    int *current_size_of_split = (int *)malloc(splits_count * sizeof(int));
    for (int i = 0; i < splits_count; i++) {
        current_size_of_split[i] = 0;
    }


    // Example:
    //                          (empty split)
    // size_of_split[i]:    3         v  2       5
    // splits looks like: [ a1 a2 a3 | | a4 a5 | a6 a7 a8 a9 a10 ]
    //                      ^            ^       ^
    //                 current_position_for_split[i] (initial state)


    // create splits array
    float *splits = (float *)malloc(n * sizeof(float)); // !!!!

    // fill splits with histogram
    histogram_fill_splits(data, n, splits, begin_position_for_split, current_size_of_split, min, max, splits_count);



    // STEP 3: Make buckets

    int buckets_count = splits_count;
    int *size_of_bucket = (int *)malloc(buckets_count * sizeof(int)); // how many buckets??

    for (int i = 0; i < buckets_count; i++) {
        size_of_bucket[i] = 0;
    }

    int *begin_position_for_bucket = (int *)malloc(buckets_count * sizeof(int));


    int bucket_index = 0;

    for (int split_index = 0; split_index < splits_count; split_index++) {

        int split_size = size_of_split[split_index];

        if (split_size > BUCKET_SIZE) {

            bucket_index++;

            // sort current split
            float *split = &splits[begin_position_for_split[split_index]];
            bucket_sort(split, split_size);

            // remember split as bucket
            begin_position_for_bucket[bucket_index] = begin_position_for_split[split_index];
            size_of_bucket[bucket_index] = size_of_split[split_index];
            bucket_index++;

        } else {
            int current_bucket_remaining_capacity = BUCKET_SIZE - size_of_bucket[bucket_index];
            if (split_size <= current_bucket_remaining_capacity) {
                // insert split to current bucket
                if (current_bucket_remaining_capacity == BUCKET_SIZE) {
                    begin_position_for_bucket[bucket_index] = begin_position_for_split[split_index];
                }
                size_of_bucket[bucket_index] += size_of_split[split_index];

            } else {
                // insert split to next bucket
                bucket_index++;
                begin_position_for_bucket[bucket_index] = begin_position_for_split[split_index];
                size_of_bucket[bucket_index] = size_of_split[split_index];
            }
        }
    }

    // determine correct buckets count
    if (size_of_bucket[bucket_index] == 0) {
        buckets_count = bucket_index;
    } else {
        buckets_count = bucket_index + 1;
    }



    // STEP 4: Sort buckets

    for (int i = 0; i < buckets_count; i++) {
        float *bucket = &splits[begin_position_for_bucket[i]];
        int bucket_size = size_of_bucket[i];
        odd_even_sort(bucket, bucket_size);
    }

    memcpy(data, splits, n * sizeof(float));

    depth_dec();
}




// =============================================================================
//                                  MAIN
// =============================================================================

bool sorted(float *data, int n) {
    for (int i = 1; i < n; i++) {
        if (data[i] < data[i - 1]) {
            return false;
        }
    }
    return true;
}


int main() {

/*
test
10
3 -5 6 7 4 1 8 10 2 5
*/

    int n = 0;
    float *data = read_data_as_plain_text(&n);
    // float *data = read_data(&n);




    // test gpu reduce

    float *data_device = NULL;
    CSC(hipMalloc((void **)&data_device, n * sizeof(float)));
    CSC(hipMemcpy(data_device, data, n * sizeof(float), hipMemcpyHostToDevice));

    float result_host = 0;

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    //
    // hipEventRecord(start, 0);

    recursive_gpu_reduce_max(data_device, n, &result_host);

    // hipEventRecord(stop, 0);
    //
    // hipEventSynchronize(stop);
    //
    // float elapsed_time;
    // hipEventElapsedTime(&elapsed_time, start, stop);
    //
    // hipEventDestroy(start);
    // hipEventDestroy(stop);
    //
    // printf("reduce result: %f\n", result_host);

    printf("time spent: %f\n", elapsed_time);





    // cpu test
    // float result = -1;
    //
    // clock_t begin, end;
    // double time_spent;
    //
    // begin = clock();
    // reduce(data, n, &result, &max_float, 0);
    // end = clock();
    //
    // time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    //
    // printf("reduce result: %f\n", result);
    // printf("time spent: %lf\n", time_spent);







    // // sorting
    // bucket_sort(data, n);
    //
    // // write_data_with_size(data, n);
    //
    // // printf("\n-------------------------------\n");
    // if (sorted(data, n)) {
    //     printf("Status: OK\n");
    //     // printf("Max recursion depth: %d\n", max_depth);
    // } else {
    //     printf("Status: WA\n");
    // }

    return 0;
}
