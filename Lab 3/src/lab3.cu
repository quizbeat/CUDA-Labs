//
//  lab3.cu
//  CUDA-Lab-3
//
//  Created by Nikita Makarov on 07/05/16.
//  Copyright © 2016 Nikita Makarov. All rights reserved.
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <limits>
#include <cfloat>
#include <math.h>
#include <time.h>
#include <fstream>

#define CSC(call) {														\
    hipError_t err = call;												\
    if(err != hipSuccess) {											\
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	\
            __FILE__, __LINE__, hipGetErrorString(err));				\
        exit(1);														\
    }																	\
} while (0)


#define MIN(a, b) (a < b ? a : b)
#define MAX(a, b) (a > b ? a : b)

#define EPS 1e-7

// gpu defined properties
#define GRID_SIZE  32
#define BLOCK_SIZE 32

// some hacks
#define LOG_NUM_BANKS 5 // for 32
#define CONFLICT_FREE_OFFSET(i) ((i) >> LOG_NUM_BANKS)


// sort properties
#define BUCKET_SIZE 1024
#define SPLIT_SIZE  512


#define INDEX_FROM_FLOAT_VALUE(value,min,max,count) (int)((value-min)/(max-min)*(count-1))






// recursion depth control
int depth = 0;
int max_depth = 0;

void depth_inc() {
    depth++;
    if (depth > max_depth) {
        max_depth = depth;
    }
}

void depth_dec() {
    depth--;
}

void print_depth_bar() {
    for (int i = 0; i < depth; i++) {
        printf("__");
    }
}




// =============================================================================
//                                   PRINT
// =============================================================================

void print_array(float *data, int size) {
    for (int i = 0; i < size; i++) {
        printf("%f ", data[i]);
    }
    printf("\n");
}

void print_subarray(float *data, int begin, int end) {
    for (int i = begin; i < end; i++) {
        printf("%f ", data[i]);
    }
    printf("\n");
}

void print_array(int *data, int size) {
    for (int i = 0; i < size; i++) {
        printf("%d ", data[i]);
    }
    printf("\n");
}

void print_int_address(int *p) {
    printf(">>>> INT   Address = %p,  Value = %d\n", p, *p);
}

void print_float_address(float *p) {
    printf(">>>> FLOAT Address = %p,  Value = %f\n", p, *p);
}




// =============================================================================
//                              DATA READ/WRITE
// =============================================================================

float *read_data(int *n) {
    fread(n, sizeof(int), 1, stdin);
    float *data = (float *)malloc(*n * sizeof(float));
    fread(data, sizeof(float), *n, stdin);
    return data;
}

float *read_data_as_plain_text(int *n) {
    scanf("%d", n);
    float *data = (float *)malloc(*n * sizeof(float));
    int count = 0;
    for (int i = 0; i < *n; i++) {
        scanf("%f", &data[i]);
        count++;
    }
    printf("data count: %d\n", count);
    return data;
}

void write_data(float *data, int n) {
    fwrite(data, sizeof(float), n, stdout);
}

void write_data_with_size(float *data, int n) {
    fwrite(&n, sizeof(int), 1, stdout);
    fwrite(data, sizeof(float), n, stdout);
}




// =============================================================================
//                                    HELPERS
// =============================================================================

float max_float(float lhs, float rhs) {
    return (lhs > rhs) ? lhs : rhs;
}

float min_float(float lhs, float rhs) {
    return (lhs < rhs) ? lhs : rhs;
}

float sum_float(float lhs, float rhs) {
    return lhs + rhs;
}

float multiply_float(float lhs, float rhs) {
    return lhs * rhs;
}

int sum_int(int lhs, int rhs) {
    return lhs + rhs;
}

// map float value to split index
int index_from_float_value(float value, float min, float max, int splits_count) {
    int index = (int)((value - min) / (max - min) * (splits_count - 1));
    return index;
}

void swap(float *lhs, float *rhs) {
    float temp = *lhs;
    *lhs = *rhs;
    *rhs = temp;
}



// =============================================================================
//                                   REDUCE
// =============================================================================

// plain cpu reduce
void reduce(float *data, int n, float *result, float (*op)(float, float), float identity) {
    *result = identity;
    for (int i = 0; i < n; i++) {
        *result = op(data[i], *result);
    }
}

void reduce_min_max(float *data, int size, float *min, float *max) {
    *min =  FLT_MAX;
    *max = -FLT_MAX;
    for (int i = 0; i < size; i++) {
        if (data[i] > *max) { *max = data[i]; }
        if (data[i] < *min) { *min = data[i]; }
    }
}



// gpu reduce
__global__ void gpuReduceSumFloat(float *data, int n, float *result) {

    __shared__ float shared_data[2048];

    int global_thread_id = 2 * blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id = threadIdx.x;

    if (global_thread_id + blockDim.x < n) {
        shared_data[thread_id] = data[global_thread_id] + data[global_thread_id + blockDim.x];
    } else if (global_thread_id < n) {
        shared_data[thread_id] = data[global_thread_id];
    } else {
        shared_data[thread_id] = 0;
    }

    __syncthreads();;

    // reduction process
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (thread_id < i) {
            shared_data[thread_id] += shared_data[thread_id + i];
        }
        __syncthreads();
    }

    // write result to global memory
    if (thread_id == 0) {
        result[blockIdx.x] = shared_data[0];
    }
}

__global__ void gpuReduceMaxFloat(float *data, int n, float *result) {

    __shared__ float shared_data[2048];

    int global_thread_id = 2 * blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id = threadIdx.x;

    if (global_thread_id + blockDim.x < n) {
        shared_data[thread_id] = MAX(data[global_thread_id], data[global_thread_id + blockDim.x]);
    } else if (global_thread_id < n) {
        shared_data[thread_id] = data[global_thread_id];
    } else {
        shared_data[thread_id] = data[0]; // just dummy
    }

    __syncthreads();;

    // reduction process
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (thread_id < i) {
            shared_data[thread_id] = MAX(shared_data[thread_id], shared_data[thread_id + i]);
        }
        __syncthreads();
    }

    // write result to global memory
    if (thread_id == 0) {
        result[blockIdx.x] = shared_data[0];
    }
}

__global__ void gpuReduceMinFloat(float *data, int n, float *result) {

    __shared__ float shared_data[2048];

    int global_thread_id = 2 * blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id = threadIdx.x;

    if (global_thread_id + blockDim.x < n) {
        shared_data[thread_id] = MIN(data[global_thread_id], data[global_thread_id + blockDim.x]);
    } else if (global_thread_id < n) {
        shared_data[thread_id] = data[global_thread_id];
    } else {
        shared_data[thread_id] = data[0]; // just dummy
    }

    __syncthreads();;

    // reduction process
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (thread_id < i) {
            shared_data[thread_id] = MIN(shared_data[thread_id], shared_data[thread_id + i]);
        }
        __syncthreads();
    }

    // write result to global memory
    if (thread_id == 0) {
        result[blockIdx.x] = shared_data[0];
    }
}

__host__ void recursive_gpu_reduce_sum(float *data_device, int n, float *result_host) {
    int gridSize = (n / 2048) + 1;
    int blockSize = 1024;

    // printf("gridSize = %d\n", gridSize);

    float *result_device = NULL;
    CSC(hipMalloc((void **)&result_device, gridSize * sizeof(float)));
    CSC(hipGetLastError());

    if (result_device == NULL) {
        printf("YOLO! NULL MEMORY DETECTED!\n");
    }

    gpuReduceSumFloat <<<gridSize, blockSize>>> (data_device, n, result_device);
    CSC(hipGetLastError());

    if (gridSize > 1) {
        recursive_gpu_reduce_sum(result_device, gridSize, result_host);
    } else {
        CSC(hipMemcpy(result_host, result_device, sizeof(float), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());
    }
    CSC(hipFree(result_device));
    CSC(hipGetLastError());
}

__host__ void recursive_gpu_reduce_max(float *data_device, int n, float *result_host) {
    int gridSize = (n / 2048) + 1;
    int blockSize = 1024;

    // printf("gridSize = %d\n", gridSize);

    float *result_device = NULL;
    CSC(hipMalloc((void **)&result_device, gridSize * sizeof(float)));
    CSC(hipGetLastError());

    if (result_device == NULL) {
        printf("YOLO! NULL MEMORY DETECTED!\n");
    }

    gpuReduceMaxFloat <<<gridSize, blockSize>>> (data_device, n, result_device);
    CSC(hipGetLastError());

    if (gridSize > 1) {
        recursive_gpu_reduce_max(result_device, gridSize, result_host);
    } else {
        CSC(hipMemcpy(result_host, result_device, sizeof(float), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());
    }
    CSC(hipFree(result_device));
    CSC(hipGetLastError());
}

__host__ void recursive_gpu_reduce_min(float *data_device, int n, float *result_host) {
    int gridSize = (n / 2048) + 1;
    int blockSize = 1024;

    // printf("gridSize = %d\n", gridSize);

    float *result_device = NULL;
    CSC(hipMalloc((void **)&result_device, gridSize * sizeof(float)));
    CSC(hipGetLastError());

    if (result_device == NULL) {
        printf("YOLO! NULL MEMORY DETECTED!\n");
    }

    printf("before reduce min: n = %d, gridSize = %d, blockSize = %d\n", n, gridSize, blockSize);
    gpuReduceMinFloat <<<gridSize, blockSize>>> (data_device, n, result_device);
    CSC(hipGetLastError());

    if (gridSize > 1) {
        recursive_gpu_reduce_min(result_device, gridSize, result_host);
    } else {
        CSC(hipMemcpy(result_host, result_device, sizeof(float), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());
    }
    CSC(hipFree(result_device));
    CSC(hipGetLastError());
}






// =============================================================================
//                                    SCAN
// =============================================================================

// plain cpu scan
// floating point numbers scan
void scan(float *data, int n, float *result_data, float (*op)(float, float), float identity, bool inclusive) {
    float result = identity;
    for (int i = 0; i < n; i++) {
        if (inclusive) {
            result = op(result, data[i]);
            result_data[i] = result;
        } else { // exclusive
            result_data[i] = result;
            result = op(result, data[i]);
        }
    }
}

// integer numbers scan
void scan(int *data, int n, int *result_data, int (*op)(int, int), int identity, bool inclusive) {
    int result = identity;
    for (int i = 0; i < n; i++) {
        if (inclusive) {
            result = op(result, data[i]);
            result_data[i] = result;
        } else { // exclusive
            result_data[i] = result;
            result = op(result, data[i]);
        }
    }
}

void scan_calculate_positions(int *position_for_split, int n, int *size_of_split) {
    for (int i = 1; i < n; i++) {
        position_for_split[i] = position_for_split[i - 1] + size_of_split[i - 1];
    }
}


// gpu scan
// Blelloch Scan
__global__ void BlellochScanSumFloat(float *data, int n, float *result) {
    __shared__ float shared_data[2 * BLOCK_SIZE];

    int thread_id = threadIdx.x;
    int offset = 1;

    shared_data[thread_id + BLOCK_SIZE] = data[thread_id + BLOCK_SIZE];

    for (int i = n / 2; i > 0; i /= 2) {
        __syncthreads();
        if (thread_id < i) {
            int ai = offset * (2 * thread_id + 1) - 1;
            int bi = offset * (2 * thread_id + 2) - 1;
            shared_data[bi] += shared_data[ai];
        }
        offset *= 2;
    }

    // clear last item
    if (thread_id == 0) {
        shared_data[n - 1] = 0; // ????
    }

    // copying and addition
    for (int i = 1; i < n; i *= 2) {
        offset /= 2;
        __syncthreads();
        if (thread_id < i) {
            int ai = offset * (2 * thread_id + 1) - 1;
            int bi = offset * (2 * thread_id + 2) - 1;
            float t = shared_data[ai];
            shared_data[ai] = shared_data[bi];
            shared_data[bi] += t;
        }
    }

    __syncthreads();

    // write result
    result[2 * thread_id] = shared_data[2 * thread_id];
    result[2 * thread_id + 1] = shared_data[2 * thread_id + 1];
}

__global__ void scan2(float *data, int n, float *result) {
    __shared__ float temp[2 * BLOCK_SIZE + CONFLICT_FREE_OFFSET(2 * BLOCK_SIZE)];

    int thread_id = threadIdx.x;
    int offset = 1;
    int ai = thread_id;
    int bi = thread_id + (n / 2);
    int offset_A = CONFLICT_FREE_OFFSET(ai);
    int offset_B = CONFLICT_FREE_OFFSET(bi);

    temp[ai + offset_A] = data[ai];
    temp[bi + offset_B] = data[bi];

    for (int d = n >> 1; d > 0; d >>= 1) {
        __syncthreads();
        if (thread_id < d) {
            int ai = offset * (2 * thread_id + 1) - 1;
            int bi = offset * (2 * thread_id + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            temp[bi] += temp[ai];
        }
        offset <<= 1;
    }

    if (thread_id == 0) {
        temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
    }

    for (int d = 1; d < n; d <<= 1) {
        offset >>= 1;
        __syncthreads();
        if (thread_id < d) {
            int ai = offset * (2 * thread_id + 1) - 1;
            int bi = offset * (2 * thread_id + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            float t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }

    __syncthreads();

    result[ai] = temp[ai + offset_A];
    result[bi] = temp[bi + offset_B];
}

__global__ void scan3(int *data, int n, int *sums, int *result) {
    __shared__ int temp[2 * BLOCK_SIZE + CONFLICT_FREE_OFFSET(2 * BLOCK_SIZE)];

    int thread_id = threadIdx.x;
    int offset = 1;
    int ai = thread_id;
    int bi = thread_id + (n / 2);  // different with Roma's code
    int offset_A = CONFLICT_FREE_OFFSET(ai);
    int offset_B = CONFLICT_FREE_OFFSET(bi);

    // printf("scan3 [ai + offset_A] = [%d]\nscan3 [bi + offset_B] = [%d]\n", ai + offset_A, bi + offset_B);

    temp[ai + offset_A] = data[ai + 2 * BLOCK_SIZE * blockIdx.x];
    temp[bi + offset_B] = data[bi + 2 * BLOCK_SIZE * blockIdx.x];

    for (int d = n >> 1; d > 0; d >>= 1) {
        __syncthreads();
        if (thread_id < d) {
            int ai = offset * (2 * thread_id + 1) - 1;
            int bi = offset * (2 * thread_id + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            temp[bi] += temp[ai];
        }
        offset <<= 1;
    }

    if (thread_id == 0) {
        int idx = n - 1 + CONFLICT_FREE_OFFSET(n - 1);
        sums[blockIdx.x] = temp[idx];
        temp[idx] = 0;
    }

    for (int d = 1; d < n; d <<= 1) {
        offset >>= 1;
        __syncthreads();
        if (thread_id < d) {
            int ai = offset * (2 * thread_id + 1) - 1;
            int bi = offset * (2 * thread_id + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }

    __syncthreads();

    offset = 2 * BLOCK_SIZE * blockIdx.x;

    // printf("scan3 indexes = (%d, %d)\n", ai + offset, bi + offset);

    result[ai + offset] = temp[ai + offset_A];
    result[bi + offset] = temp[bi + offset_B];
}

__global__ void scanDistribute(int *data, int *sums) {
    int idx = threadIdx.x + blockIdx.x * 2 * BLOCK_SIZE;
    // printf("scanDistribute index = %d\n", idx);
    data[idx] += sums[blockIdx.x];
}

__host__ void recursive_gpu_scan(int *data, int n, int *result) {

    printf("--- recursive_gpu_scan (data size = %d)\n", n);

    int numBlocks = n / (2 * BLOCK_SIZE) + 1;
    // if (numBlocks < 1) {
    //     numBlocks = 1;
    // }
    printf("numBlocks = %d\n", numBlocks);

    int *sums  = NULL;
    int *sums2 = NULL;

    CSC(hipMalloc((void **)&sums, numBlocks * sizeof(int)));
    CSC(hipGetLastError());

    CSC(hipMalloc((void **)&sums2, numBlocks * sizeof(int)));
    CSC(hipGetLastError());


    // CSC(cudaMemset(sums, 0.f, numBlocks * sizeof(float)));
    // CSC(cudaMemset(sums2, 0.f, numBlocks * sizeof(float)));

    dim3 threads(BLOCK_SIZE, 1, 1);
    dim3 blocks(numBlocks, 1, 1);

    scan3 <<<blocks, threads>>> (data, 2 * BLOCK_SIZE, sums, result);
    CSC(hipGetLastError());


    // float *result_host = (float *)malloc(n * sizeof(float));
    // CSC(cudaMemcpy(result_host, result, n * sizeof(float), cudaMemcpyDeviceToHost));
    // printf("result after scan3\n");
    // print_array(result_host, n);


    if (n >= 2 * BLOCK_SIZE) {
        // printf("%d >= 2 * %d\n", n, BLOCK_SIZE);
        recursive_gpu_scan(sums, numBlocks, sums2);
        CSC(hipGetLastError());
    } else {
        CSC(hipMemcpy(sums2, sums, numBlocks * sizeof(int), hipMemcpyDeviceToDevice));
        CSC(hipGetLastError());
    }

    if (numBlocks > 1) {
        threads = dim3(2 * BLOCK_SIZE, 1, 1);
        blocks = dim3(numBlocks - 1, 1, 1);

        printf("before distribute: blocks = %d, threads = %d\n", blocks.x, threads.x);
        scanDistribute <<<blocks, threads>>> (result + (2 * BLOCK_SIZE), sums2 + 1);
        CSC(hipGetLastError());
    }

    // CSC(cudaMemcpy(result_host, result, n * sizeof(float), cudaMemcpyDeviceToHost));
    // printf("result after scanDistribute\n");
    // print_array(result_host, n);

    hipFree(sums);
    CSC(hipGetLastError());

    hipFree(sums2);
    CSC(hipGetLastError());
}


// =============================================================================
//                                  HISTOGRAM
// =============================================================================

void histogram(float *data, int n, int *result, int op(float value, float min, float max, int count), float min, float max, int count) {
    for (int i = 0; i < n; i++) {
        int index = op(data[i], min, max, count);
        result[index]++;
    }
}

void histogram_splits_sizes(float *data, int n, int *size_of_split, float min, float max, int splits_count) {
    for (int i = 0; i < n; i++) {
        int index = index_from_float_value(data[i], min, max, splits_count);
        size_of_split[index]++;
    }
}

void histogram_fill_splits(float *data, int n, float *splits, int *begin_postion_for_split, int *current_size_of_split, float min, float max, int splits_count) {
    for (int i = 0; i < n; i++) {
        float value = data[i];
        int split_index = index_from_float_value(value, min, max, splits_count);
        int insert_position = begin_postion_for_split[split_index] + current_size_of_split[split_index];
        splits[insert_position] = value;
        current_size_of_split[split_index]++;
    }
}


// gpu histogram
__global__ void gpuHistogramCalculateSplitsSizes(float *data, int n, int *result, float min, float max, int splits_count) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    for ( ; index < n; index += offset) {
        int insertion_index = INDEX_FROM_FLOAT_VALUE(data[index], min, max, splits_count);
        atomicAdd(&(result[insertion_index]), 1);
    }
}

__global__ void gpuHistogramFillSplits(float *data_device, int n, float *splits_device,
                                       int *begin_position_for_split_device,
                                       unsigned int *current_size_of_split_device,
                                       float min, float max, int splits_count)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    for ( ; index < n; index += offset) {
        float value = data_device[index];
        int split_index = INDEX_FROM_FLOAT_VALUE(value, min, max, splits_count); // can I call this function from GPU?..
        int current_size_of_split = atomicInc(&(current_size_of_split_device[split_index]), 1); // WARNING! POTENTIAL ERROR!
        int insert_position = begin_position_for_split_device[split_index] + current_size_of_split;
        splits_device[insert_position] = value;
    }
}



// =============================================================================
//                                   SORT
// =============================================================================

void odd_even_sort(float *data, int size) {
    for (int i = 0; i < size; i++) {
        for (int j = i & 1; j < size - 1; j += 2) {
            if (data[j] > data[j + 1]) {
                swap(&data[j], &data[j + 1]);
            }
        }
    }
}

void odd_even_sort_range(float *data, int begin, int end) {
    int size = end - begin;
    for (int i = 0; i < size; i++) {
        for (int j = i & 1; j < size - 1; j += 2) {
            if (data[j + begin] > data[j + begin + 1]) {
                swap(&data[j + begin], &data[j + begin + 1]);
            }
        }
    }
}

void bucket_sort(float *data, int n) {

    depth_inc();

    float min = FLT_MAX;
    float max = -FLT_MAX;

    reduce(data, n, &min, &min_float,  FLT_MAX);
    reduce(data, n, &max, &max_float, -FLT_MAX);

    if (fabs(min - max) < EPS) { // data already sorted
        depth_dec();
        return;
    }


    // STEP 2: Make splits

    // calculate splits count
    int splits_count = n / SPLIT_SIZE + 1;


    // create array for splits sizes
    int *size_of_split = (int *)malloc(splits_count * sizeof(int));
    for (int i = 0; i < splits_count; i++) {
        size_of_split[i] = 0;
    }


    // perform histogram to calculate splits sizes
    histogram(data, n, size_of_split, &index_from_float_value, min, max, splits_count);


    // calculate splits begin position
    int *begin_position_for_split = (int *)malloc(splits_count * sizeof(int));
    scan(size_of_split, splits_count, begin_position_for_split, &sum_int, 0, false);


    int *current_size_of_split = (int *)malloc(splits_count * sizeof(int));
    for (int i = 0; i < splits_count; i++) {
        current_size_of_split[i] = 0;
    }


    // Example:
    //                          (empty split)
    // size_of_split[i]:    3         v  2       5
    // splits looks like: [ a1 a2 a3 | | a4 a5 | a6 a7 a8 a9 a10 ]
    //                      ^            ^       ^
    //                 current_position_for_split[i] (initial state)


    // create splits array
    float *splits = (float *)malloc(n * sizeof(float)); // !!!!

    // fill splits with histogram
    histogram_fill_splits(data, n, splits, begin_position_for_split, current_size_of_split, min, max, splits_count);



    // STEP 3: Make buckets

    int buckets_count = splits_count;
    int *size_of_bucket = (int *)malloc(buckets_count * sizeof(int)); // how many buckets??

    for (int i = 0; i < buckets_count; i++) {
        size_of_bucket[i] = 0;
    }

    int *begin_position_for_bucket = (int *)malloc(buckets_count * sizeof(int));


    int bucket_index = 0;

    for (int split_index = 0; split_index < splits_count; split_index++) {

        int split_size = size_of_split[split_index];

        if (split_size > BUCKET_SIZE) {

            bucket_index++;

            // sort current split
            float *split = &splits[begin_position_for_split[split_index]];
            bucket_sort(split, split_size);

            // remember split as bucket
            begin_position_for_bucket[bucket_index] = begin_position_for_split[split_index];
            size_of_bucket[bucket_index] = size_of_split[split_index];
            bucket_index++;

        } else {
            int current_bucket_remaining_capacity = BUCKET_SIZE - size_of_bucket[bucket_index];
            if (split_size <= current_bucket_remaining_capacity) {
                // insert split to current bucket
                if (current_bucket_remaining_capacity == BUCKET_SIZE) {
                    begin_position_for_bucket[bucket_index] = begin_position_for_split[split_index];
                }
                size_of_bucket[bucket_index] += size_of_split[split_index];

            } else {
                // insert split to next bucket
                bucket_index++;
                begin_position_for_bucket[bucket_index] = begin_position_for_split[split_index];
                size_of_bucket[bucket_index] = size_of_split[split_index];
            }
        }
    }

    // determine correct buckets count
    if (size_of_bucket[bucket_index] == 0) {
        buckets_count = bucket_index;
    } else {
        buckets_count = bucket_index + 1;
    }



    // STEP 4: Sort buckets

    for (int i = 0; i < buckets_count; i++) {
        float *bucket = &splits[begin_position_for_bucket[i]];
        int bucket_size = size_of_bucket[i];
        odd_even_sort(bucket, bucket_size);
    }

    memcpy(data, splits, n * sizeof(float));

    depth_dec();
}




//
// gpu_bucket_sort description:
//      data -- initial array allocated for GPU usage;
//         n -- amount of items in data array;
//    result -- sorted data array allocated for CPU usage.
//
__host__ void gpu_bucket_sort(float *data_device, int n) {

    printf("BEGIN SORT\n");

    // find min data value
    float min = FLT_MAX;
    recursive_gpu_reduce_min(data_device, n, &min);
    CSC(hipGetLastError());

    // find max data value
    float max = -FLT_MAX;
    recursive_gpu_reduce_max(data_device, n, &max);
    CSC(hipGetLastError());

    // check for already sorted array
    if (fabs(min - max) < EPS) {
        return;
    }

    // make splits
    int splits_count = n / SPLIT_SIZE + 1;

    int *size_of_split = (int *)malloc(splits_count * sizeof(int));
    for (int i = 0; i < splits_count; i++) {
        size_of_split[i] = 0;
    }

    // create size_of_split on gpu
    int *size_of_split_device = NULL;
    CSC(hipMalloc((void **)&size_of_split_device, splits_count * sizeof(int)));
    CSC(hipGetLastError());

    // calculate splits sizes with histogram
    gpuHistogramCalculateSplitsSizes <<<GRID_SIZE, BLOCK_SIZE>>> (data_device, n, size_of_split_device, min, max, splits_count);
    CSC(hipGetLastError());

    // calculate splits begin position with scan
    int *begin_position_for_split_device = NULL;
    CSC(hipMalloc((void **)&begin_position_for_split_device, splits_count * sizeof(int)));
    CSC(hipGetLastError());

    recursive_gpu_scan(size_of_split_device, n, begin_position_for_split_device);
    CSC(hipGetLastError());

    // int *current_size_of_split = (int *)malloc(splits_count * sizeof(int));
    // for (int i = 0; i < splits_count; i++) {
    //     current_size_of_split[i] = 0;
    // }

    unsigned int *current_size_of_split_device = NULL;
    CSC(hipMalloc((void **)&current_size_of_split_device, splits_count * sizeof(unsigned int)));
    CSC(hipGetLastError());

    CSC(hipMemset(current_size_of_split_device, 0, splits_count * sizeof(unsigned int)));
    CSC(hipGetLastError());

    // create splits array
    float *splits_device = NULL;
    CSC(hipMalloc((void **)&splits_device, n * sizeof(float)));
    CSC(hipGetLastError());

    // fill splits with histogram
    gpuHistogramFillSplits <<<GRID_SIZE, BLOCK_SIZE>>> (data_device, n, splits_device,
                                                        begin_position_for_split_device,
                                                        current_size_of_split_device,
                                                        min, max, splits_count);
    CSC(hipGetLastError());

    // make buckets
    int buckets_count = splits_count;
    int *size_of_bucket = (int *)malloc(buckets_count * sizeof(int));
    memset(size_of_bucket, 0, buckets_count * sizeof(int));

    int *begin_position_for_bucket = (int *)malloc(buckets_count * sizeof(int));

    int bucket_index = 0;

    for (int split_index = 0; split_index < splits_count; split_index++) {

        int split_size = 0;
        CSC(hipMemcpy(&split_size, &(size_of_split_device[split_index]), sizeof(int), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());

        int split_begin_position = 0;
        CSC(hipMemcpy(&split_begin_position, &(begin_position_for_split_device[split_index]), sizeof(int), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());

        printf("split_size = %d\n", split_size);
        printf("split_begin_position = %d\n", split_begin_position);

        if (split_size > BUCKET_SIZE) {

            printf(">>>>>>>>>>>>> YEP\n");

            bucket_index++;

            // sort current split
            float *split = &(splits_device[split_begin_position]); // gpu pointer
            gpu_bucket_sort(split, split_size);

            // remember split as bucket
            begin_position_for_bucket[bucket_index] = split_begin_position; // ????
            // size_of_bucket[bucket_index] = split_size; // ????
            size_of_bucket[bucket_index] = -1; // -1 indicates that bucket already sorted
            bucket_index++;

        } else {

            int current_bucket_remaining_capacity = BUCKET_SIZE - size_of_bucket[bucket_index];
            if (split_size <= current_bucket_remaining_capacity) {
                // insert split to current bucket
                if (current_bucket_remaining_capacity == BUCKET_SIZE) {
                    begin_position_for_bucket[bucket_index] = split_begin_position;
                }
                size_of_bucket[bucket_index] += split_size;

            } else {
                // insert split to next bucket
                bucket_index++;
                begin_position_for_bucket[bucket_index] = split_begin_position;
                size_of_bucket[bucket_index] = split_size;
            }
        }
    }

    // determine correct buckets count
    if (size_of_bucket[bucket_index] == 0) {
        buckets_count = bucket_index;
    } else {
        buckets_count = bucket_index + 1;
    }

    // sort buckets

    for (int i = 0; i < buckets_count; i++) {
        int bucket_size = size_of_bucket[i];
        if (bucket_size == -1) { // already sorted
            continue;
        }
        float *bucket = (float *)malloc(bucket_size * sizeof(float));
        int bucket_begin_position = begin_position_for_bucket[i];

        CSC(hipMemcpy(bucket, &(splits_device[bucket_begin_position]), bucket_size * sizeof(float), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());

        odd_even_sort(bucket, bucket_size);

        CSC(hipMemcpy(&(splits_device[bucket_begin_position]), bucket, bucket_size * sizeof(float), hipMemcpyHostToDevice));
        CSC(hipGetLastError());
    }

    CSC(hipMemcpy(data_device, splits_device, n * sizeof(float), hipMemcpyDeviceToDevice));

    printf("END SORT\n");

}


__host__ void __bucket_sort(float *data, int n) {
    // prepare device data
    float *data_device = NULL;
    CSC(hipMalloc((void **)&data_device, n * sizeof(float)));
    CSC(hipMemcpy(data_device, data, n * sizeof(float), hipMemcpyHostToDevice));
    CSC(hipGetLastError());

    gpu_bucket_sort(data_device, n);

    CSC(hipMemcpy(data, data_device, n * sizeof(float), hipMemcpyDeviceToHost));
    CSC(hipGetLastError());
}


// =============================================================================
//                                  MAIN
// =============================================================================

bool sorted(float *data, int n) {
    for (int i = 1; i < n; i++) {
        if (data[i] < data[i - 1]) {
            return false;
        }
    }
    return true;
}


int main() {

/*
test
10
3 -5 6 7 4 1 8 10 2 5
*/

    int n = 0;
    float *data = read_data_as_plain_text(&n);
    // float *data = read_data(&n);

    __bucket_sort(data, n);

    print_array(data, n);

    return 0;
}
