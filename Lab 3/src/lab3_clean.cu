//
//  lab3.cu
//  CUDA-Lab-3
//
//  Created by Nikita Makarov on 07/05/16.
//  Copyright © 2016 Nikita Makarov. All rights reserved.
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <limits>
#include <cfloat>
#include <math.h>
#include <time.h>
#include <fstream>


#define CSC(call) {														\
    hipError_t err = call;												\
    if(err != hipSuccess) {											\
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	\
            __FILE__, __LINE__, hipGetErrorString(err));				\
        exit(1);														\
    }																	\
} while (0)


#define MIN(a, b) (a < b ? a : b)
#define MAX(a, b) (a > b ? a : b)
#define INDEX_FROM_FLOAT_VALUE(value,min,max,count) (int)((value-min)/(max-min)*(count-1))

#define EPS 1e-7

// GPU properties
#define GRID_SIZE_REDUCE  1024
#define BLOCK_SIZE_REDUCE 1024

#define GRID_SIZE_SCAN  256
#define BLOCK_SIZE_SCAN 32

#define GRID_SIZE_HISTOGRAM  32
#define BLOCK_SIZE_HISTOGRAM 32

#define GRID_SIZE_SORT  1024
#define BLOCK_SIZE_SORT 1024

// some hacks
#define LOG_NUM_BANKS 5
#define CONFLICT_FREE_OFFSET(i) ((i) >> LOG_NUM_BANKS)

// sort properties
#define BUCKET_SIZE 1024
#define SPLIT_SIZE  32


// recursion depth control
int depth = 0;
int max_depth = 0;

void depth_inc() {
    depth++;
    if (depth > max_depth) {
        max_depth = depth;
    }
}

void depth_dec() {
    depth--;
}

void print_depth_bar() {
    for (int i = 0; i < depth; i++) {
        printf("__");
    }
}

void print_depth_space() {
    for (int i = 0; i < depth; i++) {
        printf("  ");
    }
}



// =============================================================================
//                                   PRINT
// =============================================================================

void print_array(float *data, int size) {
    for (int i = 0; i < size; i++) {
        printf("%f ", data[i]);
    }
    printf("\n");
}

void print_array(int *data, int size) {
    for (int i = 0; i < size; i++) {
        printf("%d ", data[i]);
    }
    printf("\n");
}



// =============================================================================
//                              DATA READ/WRITE
// =============================================================================

float *read_data(int *n) {
    fread(n, sizeof(int), 1, stdin);
    float *data = (float *)malloc(*n * sizeof(float));
    fread(data, sizeof(float), *n, stdin);
    return data;
}

float *read_data_as_plain_text(int *n) {
    scanf("%d", n);
    float *data = (float *)malloc(*n * sizeof(float));
    int count = 0;
    for (int i = 0; i < *n; i++) {
        scanf("%f", &data[i]);
        count++;
    }
#ifdef DEBUG
    printf("data count: %d\n", count);
#endif

    return data;
}

void write_data(float *data, int n) {
    fwrite(data, sizeof(float), n, stdout);
}

void write_data_with_size(float *data, int n) {
    fwrite(&n, sizeof(int), 1, stdout);
    fwrite(data, sizeof(float), n, stdout);
}



// =============================================================================
//                                   REDUCE
// =============================================================================

__global__ void gpuReduceMaxFloat(float *data, int n, float *result) {

    __shared__ float shared_data[2 * BLOCK_SIZE_REDUCE]; // ok

    int global_thread_id = 2 * blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id = threadIdx.x;

    if (global_thread_id + blockDim.x < n) {
        shared_data[thread_id] = MAX(data[global_thread_id], data[global_thread_id + blockDim.x]);
    } else if (global_thread_id < n) {
        shared_data[thread_id] = data[global_thread_id];
    } else {
        shared_data[thread_id] = data[0]; // just dummy
    }

    __syncthreads();

    // reduction process
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (thread_id < i) {
            shared_data[thread_id] = MAX(shared_data[thread_id], shared_data[thread_id + i]);
        }
        __syncthreads();
    }

    // write result to global memory
    if (thread_id == 0) {
        result[blockIdx.x] = shared_data[0];
    }
}

__global__ void gpuReduceMinFloat(float *data, int n, float *result) {

    __shared__ float shared_data[2 * BLOCK_SIZE_REDUCE]; // ok

    int global_thread_id = 2 * blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id = threadIdx.x;

    if (global_thread_id + blockDim.x < n) {
        shared_data[thread_id] = MIN(data[global_thread_id], data[global_thread_id + blockDim.x]);
    } else if (global_thread_id < n) {
        shared_data[thread_id] = data[global_thread_id];
    } else {
        shared_data[thread_id] = data[0]; // just dummy
    }

    __syncthreads();

    // reduction process
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (thread_id < i) {
            shared_data[thread_id] = MIN(shared_data[thread_id], shared_data[thread_id + i]);
        }
        __syncthreads();
    }

    // write result to global memory
    if (thread_id == 0) {
        result[blockIdx.x] = shared_data[0];
    }
}

__host__ void recursive_gpu_reduce_max(float *data_device, int n, float *result_host) {
    int gridSize = (n / (2 * BLOCK_SIZE_REDUCE)) + 1;
    // int blockSize = 1024;

    float *result_device = NULL;
    CSC(hipMalloc((void **)&result_device, gridSize * sizeof(float)));
    CSC(hipGetLastError());

    if (result_device == NULL) {
        printf("YOLO! NULL MEMORY DETECTED!\n");
    }

    gpuReduceMaxFloat <<<gridSize, BLOCK_SIZE_REDUCE>>> (data_device, n, result_device);
    CSC(hipGetLastError());

    if (gridSize > 1) {
        // reduce last sums
        recursive_gpu_reduce_max(result_device, gridSize, result_host);
    } else {
        CSC(hipMemcpy(result_host, result_device, sizeof(float), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());
    }

    CSC(hipFree(result_device));
    CSC(hipGetLastError());
}

__host__ void recursive_gpu_reduce_min(float *data_device, int n, float *result_host) {
    int gridSize = (n / (2 * BLOCK_SIZE_REDUCE)) + 1;
    // int blockSize = 1024;

    float *result_device = NULL;
    CSC(hipMalloc((void **)&result_device, gridSize * sizeof(float)));
    CSC(hipGetLastError());

    if (result_device == NULL) {
        print_depth_space();
        printf("YOLO! NULL MEMORY DETECTED!\n");
    }

#ifdef DEBUG
    print_depth_space();
    printf("before reduce min: n = %d, gridSize = %d, blockSize = %d\n", n, gridSize, BLOCK_SIZE_REDUCE);
#endif

    gpuReduceMinFloat <<<gridSize, BLOCK_SIZE_REDUCE>>> (data_device, n, result_device);
    CSC(hipGetLastError());

    if (gridSize > 1) {
        // reduce last sums
        recursive_gpu_reduce_min(result_device, gridSize, result_host);
    } else {
        CSC(hipMemcpy(result_host, result_device, sizeof(float), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());
    }

    CSC(hipFree(result_device));
    CSC(hipGetLastError());
}



// =============================================================================
//                                    SCAN
// =============================================================================

__global__ void gpuScan(int *data, int n, int *sums, int *result) {
    __shared__ int shared_data[2 * BLOCK_SIZE_SCAN + CONFLICT_FREE_OFFSET(2 * BLOCK_SIZE_SCAN)];

    int thread_id = threadIdx.x;
    int offset = 1;
    int ai = thread_id;
    int bi = thread_id + (n / 2);
    int offset_A = CONFLICT_FREE_OFFSET(ai);
    int offset_B = CONFLICT_FREE_OFFSET(bi);

    shared_data[ai + offset_A] = data[ai + 2 * BLOCK_SIZE_SCAN * blockIdx.x];
    shared_data[bi + offset_B] = data[bi + 2 * BLOCK_SIZE_SCAN * blockIdx.x];

    for (int index = n / 2; index > 0; index /= 2) {
        __syncthreads();
        if (thread_id < index) {
            int ai = offset * (2 * thread_id + 1) - 1;
            int bi = offset * (2 * thread_id + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            shared_data[bi] += shared_data[ai];
        }
        offset *= 2;
    }

    if (thread_id == 0) {
        int idx = n - 1 + CONFLICT_FREE_OFFSET(n - 1);
        sums[blockIdx.x] = shared_data[idx];
        shared_data[idx] = 0;
    }

    for (int index = 1; index < n; index *= 2) {
        offset /= 2;
        __syncthreads();
        if (thread_id < index) {
            int ai = offset * (2 * thread_id + 1) - 1;
            int bi = offset * (2 * thread_id + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            int t = shared_data[ai];
            shared_data[ai] = shared_data[bi];
            shared_data[bi] += t;
        }
    }

    __syncthreads();

    offset = 2 * BLOCK_SIZE_SCAN * blockIdx.x;

    result[ai + offset] = shared_data[ai + offset_A];
    result[bi + offset] = shared_data[bi + offset_B];
}

__global__ void scanDistribute(int *data, int *sums) {
    int idx = threadIdx.x + blockIdx.x * 2 * BLOCK_SIZE_SCAN;
    // printf("scanDistribute index = %d\n", idx);
    data[idx] += sums[blockIdx.x];
}

__host__ void recursive_gpu_scan(int *data, int n, int *result) {

#ifdef DEBUG
    print_depth_space();
    printf("recursive_gpu_scan (data size = %d)\n", n);
#endif

    int numBlocks = n / (2 * BLOCK_SIZE_SCAN) + 1; // 2 * 512

#ifdef DEBUG
    print_depth_space();
    printf("numBlocks = %d\n", numBlocks);
#endif

    int *sums  = NULL;
    int *sums2 = NULL;

    CSC(hipMalloc((void **)&sums, numBlocks * sizeof(int))); // n / 1024 + 1
    CSC(hipGetLastError());

    CSC(hipMalloc((void **)&sums2, numBlocks * sizeof(int))); // n / 1024 + 1
    CSC(hipGetLastError());

    dim3 threads(BLOCK_SIZE_SCAN, 1, 1);
    dim3 blocks(numBlocks, 1, 1);

    //       n/1024+1   512                     1024
    gpuScan <<<blocks, threads>>> (data, 2 * BLOCK_SIZE_SCAN, sums, result);
    CSC(hipGetLastError());

            //     1024
    if (n >= 2 * BLOCK_SIZE_SCAN) {
        //                       n/1024+1
        recursive_gpu_scan(sums, numBlocks, sums2);
        CSC(hipGetLastError());
    } else {
        CSC(hipMemcpy(sums2, sums, numBlocks * sizeof(int), hipMemcpyDeviceToDevice));
        CSC(hipGetLastError());
    }

    if (numBlocks > 1) {
        threads = dim3(2 * BLOCK_SIZE_SCAN, 1, 1);
        blocks = dim3(numBlocks - 1, 1, 1);

#ifdef DEBUG
        print_depth_space();
        printf("before distribute: blocks = %d, threads = %d\n", blocks.x, threads.x);
#endif

        scanDistribute <<<blocks, threads>>> (result + (2 * BLOCK_SIZE_SCAN), sums2 + 1);
        CSC(hipGetLastError());
    }

    if (sums == NULL) {
        printf("sums is NULL\n");
    }

    if (sums2 == NULL) {
        printf("sums2 is NULL\n");
    }

    hipFree(sums);
    CSC(hipGetLastError());

    hipFree(sums2);
    CSC(hipGetLastError());
}



// =============================================================================
//                                  HISTOGRAM
// =============================================================================

// gpu histogram
__global__ void gpuHistogramCalculateSplitsSizes(float *data, int n, int *result, float min, float max, int splits_count) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    for ( ; index < n; index += offset) {
        int insertion_index = INDEX_FROM_FLOAT_VALUE(data[index], min, max, splits_count);
        atomicAdd(&(result[insertion_index]), 1);
    }
}

__global__ void gpuHistogramFillSplits(float *data_device, int n, float *splits_device,
                                       int *begin_position_for_split_device,
                                       unsigned int *current_size_of_split_device,
                                       float min, float max, int splits_count)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    for ( ; index < n; index += offset) {
        float value = data_device[index];
        int split_index = INDEX_FROM_FLOAT_VALUE(value, min, max, splits_count);
        int current_size_of_split = atomicAdd(&(current_size_of_split_device[split_index]), 1);
        int insert_position = begin_position_for_split_device[split_index] + current_size_of_split;
        splits_device[insert_position] = value;
#ifdef DEBUG
        // printf("Block_id = %d, thread_id = %d -- insert value [%f] from index [%d] to splits index [%d]\n", blockIdx.x, threadIdx.x, value, index, insert_position);
#endif
    }
}



// =============================================================================
//                                   SORT
// =============================================================================

void swap(float *lhs, float *rhs) {
    float temp = *lhs;
    *lhs = *rhs;
    *rhs = temp;
}

// gpu sort for single thread
__global__ void gpuOddEvenSort(float *data, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = i & 1; j < n - 1; j += 2) {
            if (data[j] > data[j + 1]) {
                float temp = data[j];
                data[j] = data[j + 1];
                data[j + 1] = temp;
            }
        }
    }
}

// multiple threads
__global__ void oddEvenSort(float *data, int n, int buckets_count, int *begin_position_for_bucket, int *size_of_bucket) {
    int bucket_index = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;

    for (int b = bucket_index; b < buckets_count; b += offset) {
        int size = size_of_bucket[bucket_index];
        if (size == -1) { // already sorted
            continue;
        }
        int begin = begin_position_for_bucket[b];
        for (int i = begin; i < begin + size; i++) {
            for (int j = i & 1; j < begin + size - 1; j += 2) {
                if (data[j] > data[j + 1]) {
                    float temp = data[j];
                    data[j] = data[j + 1];
                    data[j + 1] = temp;
                }
            }
        }
    }
}




//
// gpu_bucket_sort description:
//  data_device -- initial array allocated for GPU usage;
//            n -- amount of items in data array;
//
__host__ void gpu_bucket_sort(float *data_device, int n) {

#ifdef DEBUG
    depth_inc();
    print_depth_space();
    printf("BEGIN SORT\n");
#endif

    // find min data value
    float min = FLT_MAX;
    recursive_gpu_reduce_min(data_device, n, &min); // find min
    CSC(hipGetLastError());

    // find max data value
    float max = -FLT_MAX;
    recursive_gpu_reduce_max(data_device, n, &max); // find max
    CSC(hipGetLastError());

#ifdef DEBUG
    print_depth_space();
    printf("MIN = %f, MAX = %f\n", min, max);
#endif

    // check for already sorted array
    if (fabs(min - max) < EPS) {
#ifdef DEBUG
        depth_dec();
#endif
        return;
    }

    // Example:
    //                          (empty split)
    // size_of_split[i]:    3         v  2       5
    // splits looks like: [ a1 a2 a3 | | a4 a5 | a6 a7 a8 a9 a10 ]
    //                      ^            ^       ^
    //                 current_position_for_split[i] (initial state)

    // make splits
    int splits_count = n / SPLIT_SIZE + 1;

#ifdef DEBUG
    print_depth_space();
    printf("splits_count = %d\n", splits_count);
#endif

    // create size_of_split on gpu
    int *size_of_split_device = NULL;
    CSC(hipMalloc((void **)&size_of_split_device, splits_count * sizeof(int)));
    CSC(hipMemset(size_of_split_device, 0, splits_count * sizeof(int)));
    CSC(hipGetLastError());

    // calculate splits sizes with histogram
    gpuHistogramCalculateSplitsSizes <<<GRID_SIZE_HISTOGRAM, BLOCK_SIZE_HISTOGRAM>>> (data_device, n, size_of_split_device, min, max, splits_count);
    CSC(hipGetLastError());

#ifdef DEBUG // check size_of_split array

    int *size_of_split = (int *)malloc(splits_count * sizeof(int));
    memset(size_of_split, 0, splits_count * sizeof(int));

    CSC(cudaMemcpy(size_of_split, size_of_split_device, splits_count * sizeof(int), cudaMemcpyDeviceToHost));
    CSC(cudaGetLastError());

    print_depth_space();
    printf("size_of_split: ");
    for (int i = 0; i < splits_count; i++) {
        printf("%d ", size_of_split[i]);
    }
    printf("\n");

    free(size_of_split);

#endif

    // calculate splits begin position with scan
    int *begin_position_for_split_device = NULL;
    CSC(hipMalloc((void **)&begin_position_for_split_device, splits_count * sizeof(int)));
    CSC(hipGetLastError());

    recursive_gpu_scan(size_of_split_device, n, begin_position_for_split_device);
    CSC(hipGetLastError());

#ifdef DEBUG // check begin_position_for_split

    int *begin_position_for_split = (int *)malloc(splits_count * sizeof(int));
    CSC(cudaMemcpy(begin_position_for_split, begin_position_for_split_device, splits_count * sizeof(int), cudaMemcpyDeviceToHost));
    CSC(cudaGetLastError());

    print_depth_space();
    printf("begin_position_for_split: ");
    for (int i = 0; i < splits_count; i++) {
        printf("%d ", begin_position_for_split[i]);
    }
    printf("\n");

    free(begin_position_for_split);

#endif

    unsigned int *current_size_of_split_device = NULL;
    CSC(hipMalloc((void **)&current_size_of_split_device, splits_count * sizeof(unsigned int)));
    CSC(hipGetLastError());

    CSC(hipMemset(current_size_of_split_device, 0, splits_count * sizeof(unsigned int)));
    CSC(hipGetLastError());

    // create splits array
    float *splits_device = NULL;
    CSC(hipMalloc((void **)&splits_device, n * sizeof(float)));
    CSC(hipGetLastError());

    // fill splits with histogram
    gpuHistogramFillSplits <<<GRID_SIZE_HISTOGRAM, BLOCK_SIZE_HISTOGRAM>>> (data_device, n, splits_device,
                                                                            begin_position_for_split_device,
                                                                            current_size_of_split_device,
                                                                            min, max, splits_count);
    CSC(hipGetLastError());


#ifdef DEBUG // check splits array

    float *splits = (float *)malloc(n * sizeof(float));
    CSC(cudaMemcpy(splits, splits_device, n * sizeof(float), cudaMemcpyDeviceToHost));
    CSC(cudaGetLastError());

    print_depth_space();
    printf("splits: ");
    for (int i = 0; i < n; i++) {
        printf("%f ", splits[i]);
    }
    printf("\n");

    free(splits);

#endif

    // make buckets
    int buckets_count = splits_count;
    int *size_of_bucket = (int *)malloc(buckets_count * sizeof(int));
    memset(size_of_bucket, 0, buckets_count * sizeof(int));

    int *begin_position_for_bucket = (int *)malloc(buckets_count * sizeof(int));

    int bucket_index = 0;

    for (int split_index = 0; split_index < splits_count; split_index++) {

        int split_size = 0;
        CSC(hipMemcpy(&split_size, &(size_of_split_device[split_index]), sizeof(int), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());

        int split_begin_position = 0;
        CSC(hipMemcpy(&split_begin_position, &(begin_position_for_split_device[split_index]), sizeof(int), hipMemcpyDeviceToHost));
        CSC(hipGetLastError());

#ifdef DEBUG
        print_depth_space();
        printf("split_size = %d\n", split_size);
        print_depth_space();
        printf("split_begin_position = %d\n", split_begin_position);
#endif

        if (split_size > BUCKET_SIZE) {

            bucket_index++;

            // sort current split
            float *split = &(splits_device[split_begin_position]); // gpu pointer
            gpu_bucket_sort(split, split_size);

            // remember split as bucket
            begin_position_for_bucket[bucket_index] = split_begin_position; // ????
            // size_of_bucket[bucket_index] = split_size; // ????
            size_of_bucket[bucket_index] = -1; // -1 indicates that bucket already sorted
            bucket_index++;

        } else {

            int current_bucket_remaining_capacity = BUCKET_SIZE - size_of_bucket[bucket_index];
            if (split_size <= current_bucket_remaining_capacity) {
                // insert split to current bucket
                if (current_bucket_remaining_capacity == BUCKET_SIZE) {
                    begin_position_for_bucket[bucket_index] = split_begin_position;
                }
                size_of_bucket[bucket_index] += split_size;

            } else {
                // insert split to next bucket
                bucket_index++;
                begin_position_for_bucket[bucket_index] = split_begin_position;
                size_of_bucket[bucket_index] = split_size;
            }
        }
    }

    // determine correct buckets count
    if (size_of_bucket[bucket_index] == 0) {
        buckets_count = bucket_index;
    } else {
        buckets_count = bucket_index + 1;
    }

    // sort buckets

    oddEvenSort <<<GRID_SIZE_SORT, BLOCK_SIZE_SORT>>> (splits_device, n, buckets_count, begin_position_for_bucket, size_of_bucket);
    CSC(hipGetLastError());


#ifdef DEBUG
    splits = (float *)malloc(n * sizeof(float));
    CSC(cudaMemcpy(splits, splits_device, n * sizeof(float), cudaMemcpyDeviceToHost));
    CSC(cudaGetLastError());
    print_depth_space();
    printf("splits: ");
    print_array(splits, n);
    free(splits);
#endif



    // for (int i = 0; i < buckets_count; i++) {
    //     int bucket_size = size_of_bucket[i];
    //     if (bucket_size == -1) { // already sorted
    //         continue;
    //     }
    //     // float *bucket = (float *)malloc(bucket_size * sizeof(float));
    //     int bucket_begin_position = begin_position_for_bucket[i];
    //
    //     gpuOddEvenSort <<<1, 1>>> (&(splits_device[bucket_begin_position]), bucket_size);
    //     CSC(cudaGetLastError());
    //
    //     // CSC(cudaMemcpy(bucket, &(splits_device[bucket_begin_position]), bucket_size * sizeof(float), cudaMemcpyDeviceToHost));
    //     // CSC(cudaGetLastError());
    //
    //     // odd_even_sort(bucket, bucket_size);
    //
    //     // CSC(cudaMemcpy(&(splits_device[bucket_begin_position]), bucket, bucket_size * sizeof(float), cudaMemcpyHostToDevice));
    //     // CSC(cudaGetLastError());
    //
    //     // free(bucket);
    // }

    CSC(hipMemcpy(data_device, splits_device, n * sizeof(float), hipMemcpyDeviceToDevice));

    CSC(hipFree(size_of_split_device));
    CSC(hipFree(splits_device));
    CSC(hipFree(begin_position_for_split_device));
    CSC(hipFree(current_size_of_split_device));
    CSC(hipGetLastError());

    free(size_of_bucket);
    free(begin_position_for_bucket);

#ifdef DEBUG
    print_depth_space();
    printf("END SORT\n");
    depth_dec();
#endif

}


//
// bucket_sort description:
//  data -- initial array allocated for CPU usage;
//     n -- amount of items in data array;
//
__host__ void bucket_sort(float *data, int n) {
    // prepare device data
    float *data_device = NULL; // pointer to gpu data
    CSC(hipMalloc((void **)&data_device, n * sizeof(float))); // alloc memory for gpu data
    CSC(hipMemcpy(data_device, data, n * sizeof(float), hipMemcpyHostToDevice)); // copy data from cpu to gpu
    CSC(hipGetLastError());

    gpu_bucket_sort(data_device, n); // start sorting

    CSC(hipMemcpy(data, data_device, n * sizeof(float), hipMemcpyDeviceToHost)); // copy data back to cpu
    CSC(hipGetLastError());
}


// =============================================================================
//                                  MAIN
// =============================================================================

bool sorted(float *data, int n) {
    for (int i = 1; i < n; i++) {
        if (data[i] < data[i - 1]) {
            return false;
        }
    }
    return true;
}


int main() {

    int n = 0;
    float *data = read_data_as_plain_text(&n); // cpu data
    // float *data = read_data(&n);

    if (n == 0) {
        free(data);
        return 0;
    }

    bucket_sort(data, n); // start sorting

    // print_array(data, n);

    if (sorted(data, n)) {
        printf("--\nStatus: OK\n");
    } else {
        printf("--\nStatus: WA\n");
    }

    free(data);

    return 0;
}
