
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
	int deviceCount;
	hipDeviceProp_t devProp;
	hipGetDeviceCount(&deviceCount);
	printf("Found %d devices\n", deviceCount);
	for(int device = 0;device < deviceCount;device++) {
		hipGetDeviceProperties(&devProp, device);
		printf("Device %d\n", device);
		printf("Compute capability      : %d.%d\n", devProp.major, devProp.minor);
		printf("Name                    : %s\n", devProp.name);
		printf("Total Global Memory     : %zu\n", devProp.totalGlobalMem);
		printf("Shared memory per block : %zu\n", devProp.sharedMemPerBlock);
		printf("Registers per block     : %d\n", devProp.regsPerBlock);
		printf("Warp size               : %d\n", devProp.warpSize);
		printf("Max threads per block   : (%d, %d, %d)\n", devProp.maxThreadsDim[0], devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);
		printf("Max block   : (%d, %d, %d)\n", devProp.maxGridSize[0], devProp.maxGridSize[1], devProp.maxGridSize[2]);
		printf("Total constant memory   : %zu\n", devProp.totalConstMem);
		printf("Multiprocessors count   : %d\n", devProp.multiProcessorCount);
	}
	return 0;
}
