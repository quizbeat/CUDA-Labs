
#include <hip/hip_runtime.h>
#include <stdio.h>


#define CSC(call) {														\
    hipError_t err = call;												\
    if(err != hipSuccess) {											\
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	\
            __FILE__, __LINE__, hipGetErrorString(err));				\
        exit(1);														\
    }																	\
} while (0)


__global__ void kernel(int *a, int *b, int *c, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = gridDim.x * blockDim.x;
	for(; idx < n; idx += offset)
		c[idx] = a[idx] + b[idx];
}

int main() {
	int i, n = 2000000;
	int *a = (int *)malloc(sizeof(int) * n);
	int *b = (int *)malloc(sizeof(int) * n);
	int *c = (int *)malloc(sizeof(int) * n);
	for(i = 0; i < n; i++)
		a[i] = b[i] = i;

	int *dev_a;
	int *dev_b;
	int *dev_c;
	hipEvent_t start, stop;
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&stop));

	CSC(hipMalloc(&dev_a, sizeof(int) * n));
	CSC(hipMalloc(&dev_b, sizeof(int) * n));
	CSC(hipMalloc(&dev_c, sizeof(int) * n));

	CSC(hipMemcpy(dev_a, a, sizeof(int) * n, hipMemcpyHostToDevice));
	CSC(hipMemcpy(dev_b, b, sizeof(int) * n, hipMemcpyHostToDevice));
	CSC(hipEventRecord(start, 0));
	//for(i = 0; i < n; i++)
	//	c[i] = a[i] + b[i];
	kernel<<<6, 256>>>(dev_a, dev_b, dev_c, n);
	CSC(hipGetLastError());
	CSC(hipEventRecord(stop, 0));
	CSC(hipEventSynchronize(stop));
	float t;
	CSC(hipEventElapsedTime(&t, start, stop));
	printf("time = %f\n", t);
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(stop));

	CSC(hipMemcpy(c, dev_c, sizeof(int) * n, hipMemcpyDeviceToHost));

	//for(i = 0; i < n; i++)
	//	printf("%d ", c[i]);
	//printf("\n");

	CSC(hipFree(dev_a));
	CSC(hipFree(dev_b));
	CSC(hipFree(dev_c));

	free(a);
	free(b);
	free(c);
	return 0;
}
