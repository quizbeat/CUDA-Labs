
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CSC(call) {														\
    hipError_t err = call;												\
    if(err != hipSuccess) {											\
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	\
            __FILE__, __LINE__, hipGetErrorString(err));				\
        exit(1);														\
    }																	\
} while (0)

__global__ void kernel(int *a, int n, int k) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	for(; idx < n; idx += offset)
		a[idx] *= k;
}

int main() {
	int i, n = 10000;
	int *a = (int *)malloc(sizeof(int) * n);
	int *dev_a;
	for(i = 0; i < n; i++)
		a[i] = 1;
	CSC(hipMalloc(&dev_a, sizeof(int) * n));
	CSC(hipMemcpy(dev_a, a, sizeof(int) * n, hipMemcpyHostToDevice));
	kernel<<<dim3(2), dim3(32)>>>(dev_a, n, 2);
	CSC(hipGetLastError());
	CSC(hipMemcpy(a, dev_a, sizeof(int) * n, hipMemcpyDeviceToHost));
	for(i = 0; i < n; i++)
		printf("%d ", a[i]);
	printf("\n");
	CSC(hipFree(dev_a));
	free(a);
	return 0;
}
